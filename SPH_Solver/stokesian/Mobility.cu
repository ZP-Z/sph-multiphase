#include "hip/hip_runtime.h"

#include "fluid_system_kern.cuh"
#include "Mobility.cuh"
#include <stdio.h>
//#include "stokesian\NeighborGrid.cuh"

__device__ float* signature;
extern __device__ ParamCarrier paramCarrier;
extern bufList	fbuf;

__inline__ __device__ __host__ int tsId(int i, int j, int k) {
	return i*9 + j*3 + k;
}

__inline__ __device__ __host__ int kroneckerDelta(int i, int j) {
	return i==j?1:0;
}

void setSignature() {
	float* ptr;
	hipMalloc(&ptr, 27*sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(signature), &ptr, sizeof(ptr));
	float val[27];
	for(int i=0; i<27; i++)
		val[i] = 0;
	val [tsId(0,1,2)] = 1;
	val [tsId(2,0,1)] = 1;
	val [tsId(1,2,0)] = 1;
	val [tsId(0,2,1)] = -1;
	val [tsId(2,1,0)] = -1;
	val [tsId(1,0,2)] = -1;
	hipMemcpy(ptr, val, sizeof(float)*27, hipMemcpyHostToDevice);
	return;
}

__device__ void PairwiseMobMatrix(cfloat3 drv, float A11[],float A1N[], float C11[], float C1N[], float B11[], float B1N[]) {
	float dr = sqrt(dot(drv, drv));
	float eu[3];
	eu[0] = drv.x/dr;
	eu[1] = drv.y/dr;
	eu[2] = drv.z/dr;

	//get XYABC mobility functions
	float X11A = 1;
	float dr3 = dr*dr*dr;
	float X1NA = 1.5/dr - 1/dr3;
	float Y11A = 1;
	float Y1NA = 0.75/dr + 0.5/dr3;
	float Y11B = 0;
	float Y1NB = -3/(4*dr*dr);
	float X11C = 0.75;
	float X1NC = 0.75/dr3;
	float Y11C = 0.75;
	float Y1NC = -0.375/dr3;

	for (int i=0; i<3; i++) {
		for (int j=0; j<3; j++) {
			A11[i*3+j] = X11A * eu[i]*eu[j] + Y11A * (kroneckerDelta(i, j)-eu[i]*eu[j]);
			A1N[i*3+j] = X1NA * eu[i]*eu[j] + Y1NA * (kroneckerDelta(i, j)-eu[i]*eu[j]);
			C11[i*3+j] = X11C * eu[i]*eu[j] + Y11C * (kroneckerDelta(i, j)-eu[i]*eu[j]);
			C1N[i*3+j] = X1NC * eu[i]*eu[j] + Y1NC * (kroneckerDelta(i, j)-eu[i]*eu[j]);
		}
	}

	for (int i=0; i<3; i++) {
		A11[i*3+i] -= 1;
		C11[i*3+i] -= 0.75;
	}

	for (int i=0; i<3; i++) {
		for (int j=0; j<3; j++) {
			float dot = 0;
			for (int k=0; k<3; k++) {
				dot = dot + signature[tsId(j, i, k)]*eu[k];
			}
			B11[i*3+j] = Y11B * dot;
			B1N[i*3+j] = -Y1NB * dot;
		}
	}


}


__global__ void MVmultiply(float* dst, float* mat, float* f,int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int Adim = 6*n;
	if (i>=Adim)
		return;
	
	float sum = 0;
	for (int j=0; j<Adim; j++) {
		sum += mat[i*Adim+j] * f[j];
	}
	dst[i] = sum;
	//printf("%d %f %f\n",i, f[i], dst[i]);
	return;
}


//calculate velocity
void getMobU(float* mat,float* f, float* u, int nsize) {
	//simply matrix multiply
	int threadnum = 256;
	int blocknum = (6*nsize-1)/threadnum + 1;

	MVmultiply<<<blocknum, threadnum>>>(u, mat, f, nsize);
	hipDeviceSynchronize();
}



//-------------------------------------------------
//-------------------------------------------------
//-------------                   -----------------
//-------------   cutoff version  -----------------
//-------------                   -----------------
//-------------------------------------------------
//-------------------------------------------------

//math functions
__inline__ __device__ __host__ void mulmv3(float* dst, float* mat, float* v) {
	float tmp[3] ={0,0,0};
	for (int i=0; i<3; i++) {
		for (int j=0; j<3; j++) {
			tmp[i] += mat[i*3+j]*v[j];
		}
	}
	dst[0] = tmp[0];
	dst[1] = tmp[1];
	dst[2] = tmp[2];
}

__device__ __host__ void mulmat(float* dst, float* a,float* b){
	for(int i=0; i<3; i++){
		for(int j=0; j<3; j++){
			dst[i*3+j]=0;
			for(int k=0;k<3;k++){
				dst[i*3+j]+=a[i*3+k]*b[k*3+j];
			}
		}
	}
}

//__global__ void Kern_getmobu_cutoff(float* pos, int nsize, float* f, float* u, sortingGrid grid) {
//	int i = blockIdx.x * blockDim.x + threadIdx.x;
//	if(i>=nsize)
//		return;
//
//	//locate i's cell
//	int id = grid.idsort[i];
//	int cell = grid.cellId[i];
//	if(cell==GRIDUNDEF)
//		return;
//
//	float3 ipos = grid.pos[id];
//	float3 dx;
//	int ncell;
//
//	int kid;
//	float3 drv;
//	int n1,n2,n3,n4;
//	float mat[9];
//	memset(mat,0,9*sizeof(float));
//	float3 utmp = make_float3(0,0,0);
//	float unity[9] = {1,0,0, 0,1,0, 0,0,1};
//	//loop through neighboring cells
//	int count = 0;
//	u[3*id] = 0;
//	u[3*id+1] = 0;
//	u[3*id+2] = 0;
//	
//	for (int j=0; j<27; j++) {
//		ncell = cell + grid.searchList[j];
//
//		count += grid.gridend[ncell] - grid.gridstart[ncell];
//		for (int k=grid.gridstart[ncell]; k<grid.gridend[ncell]; k++) {
//			kid = grid.idsort[k];
//			if(kid != id){
//				drv = grid.pos[kid] - ipos;
//				float A11[9], A1N[9], C11[9], C1N[9], B11[9], B1N[9];
//
//				PairwiseMobMatrix(drv, A11, A1N, C11, C1N, B11, B1N);
//				mulmv3((float*)&utmp, A1N, f+3*kid);
//				u[3*id] += utmp.x;
//				u[3*id+1] += utmp.y;
//				u[3*id+2] += utmp.z;
//
//				/*if (id==0) {
//					printf("%d %d %f %f %f\n",id, kid, grid.pos[kid].x, grid.pos[kid].y, grid.pos[kid].z);
//				}*/
//			}
//			else {
//				mulmv3((float*)&utmp, unity, f+3*kid);
//				u[3*id] += utmp.x;
//				u[3*id+1] += utmp.y;
//				u[3*id+2] += utmp.z;
//			}
//		}
//		
//	}
//	//printf("%d %d\n", id, count);
//}



__global__ void Kern_getmobu_walkthrough( stokesianBufList buflist, int nsize) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=nsize)
		return;
	
	int id = i;
	cfloat3 ipos = buflist.dispBuffer[id].pos;
	int kid;
	cfloat3 drv;

	cfloat3 utmp(0, 0, 0);
	float unity[9] ={1,0,0, 0,1,0, 0,0,1};
	float force[3];
	

	//buflist.cuUnew[id].Set(0,0,0);
	buflist.cuOmega[id].Set(0,0,0);
	
	
	for (int k=0; k<nsize; k++) {
		kid = k;
		if (kid != id) {
			drv = buflist.dispBuffer[kid].pos - ipos;
			
			
			float A11[9], A1N[9], C11[9], C1N[9], B11[9], B1N[9];

			PairwiseMobMatrix(drv, A11, A1N, C11, C1N, B11, B1N);

			//get velocity
			force[0] = buflist.cuForce[kid].x;
			force[1] = buflist.cuForce[kid].y;
			force[2] = buflist.cuForce[kid].z;

			mulmv3((float*)&utmp, A1N, force);
			buflist.cuUnew[id] += utmp;

			//get angle velocity
			mulmv3((float*)&utmp, B1N, force);
			buflist.cuOmega[id] += utmp;
		}
		else {
			mulmv3((float*)&utmp, unity, (float*) &buflist.cuForce[kid]);
			buflist.cuUnew[id] += utmp;
		}
	}
	
	buflist.dispBuffer[id].pos += buflist.cuUnew[id] * buflist.dt;
}

__global__ void Kern_getForce(stokesianBufList buflist, int nsize){
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=nsize)
		return;

	int id = i;
	cfloat3 ipos = buflist.dispBuffer[id].pos;
	int kid;
	cfloat3 drv;

	
	int count = 0;
	for (int k=0; k<nsize; k++) {
		kid = k;
		if (kid != id) {
			drv = buflist.dispBuffer[kid].pos - ipos;

			//near-field interactions between particles
			float dr = sqrt(dot(drv, drv));
			
			//splitting
			
			float actdist = 1.5;
			if (dr < actdist) {
				float fac = 5*(exp(actdist/dr)-2.7128*dr/actdist);
				cfloat3 intforce = drv * (-1) * fac;
				buflist.cuForce[i] += intforce;
			}

			//surface tension
			//float actdist2 = 4;
			//if (dr<actdist2) {
			//	float fac = 10*cos(1.5*3.14159*dr/actdist2);
			//	if(fac>0)
			//		fac *= 30;
			//	//printf("%f %f %f %f %f %f\n",drv.x, drv.y, drv.z, dr, fac);
			//	float3 intforce = -fac * drv/dr/dr;
			//	f[i*3+0] += intforce.x;
			//	f[i*3+1] += intforce.y;
			//	f[i*3+2] += intforce.z;
			//}
		}
	}
}

//void getMobU_cutoff(float* pos, int nsize, float* f, float* u, sortingGrid grid) {
//	int threadnum = 256;
//	int blocknum = (nsize-1)/256+1;
//
//	//Kern_getmobu_cutoff<<<blocknum,threadnum>>>(pos, nsize, f, u, grid);
//	Kern_getmobu_walkthrough<<<blocknum, threadnum>>>(pos, nsize, f, u);
//	hipDeviceSynchronize();
//	
//	return;
//}

__device__ __inline__ float getWeight(cfloat3 pos, cint3 idvec, float cellsize) {
	cfloat3 dx = pos - (cfloat3)idvec * cellsize;
	return (1 - abs(dx.x)/cellsize) * (1 - abs(dx.y)/cellsize) * (1 - abs(dx.z)/cellsize);
}

__device__ __inline__ int getId(cint3 idvec) {
	return idvec.y*paramCarrier.mpmRes.x*paramCarrier.mpmRes.z + idvec.z*paramCarrier.mpmRes.x + idvec.x;
}

__global__ void Kern_interpolateVelocity(stokesianBufList buflist, int pnum, bufList fbuf) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=pnum)
		return;

	cfloat3 ipos = buflist.dispBuffer[i].pos;
	cint3 mpmres = paramCarrier.mpmRes;
	float cellsize = paramCarrier.mpmcellsize;
	
	ipos = ipos - paramCarrier.mpmXmin;
	cfloat3 idvecf = ipos / paramCarrier.mpmcellsize;
	cint3 idvec(idvecf.x, idvecf.y, idvecf.z);
	cfloat3 dx;
	float weight;
	cfloat3 velsum(0,0,0);

	
	cint3 tmp3;
	tmp3 = idvec + cint3(0, 0, 0);
	velsum  += fbuf.mpmVel[ getId(tmp3) ] * getWeight(ipos, tmp3, cellsize);
	
	tmp3 = idvec + cint3(1, 0, 0);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(0, 1, 0);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(0, 0, 1);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(1, 1, 0);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(1, 0, 1);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(0, 1, 1);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	tmp3 = idvec + cint3(1, 1, 1);
	velsum  += fbuf.mpmVel[getId(tmp3)] * getWeight(ipos, tmp3, cellsize);

	//printf("%f %f %f\n",velsum.x, velsum.y,velsum.z);
	buflist.cuUnew[i] = velsum / paramCarrier.simscale;
}


void getMobU_walkthrough(stokesianBufList buflist, int pnum) {
	int threadnum = 256;
	int blocknum = (pnum-1)/256+1;

	Kern_getForce<<<blocknum, threadnum>>>(buflist, pnum);
	hipDeviceSynchronize();

	Kern_interpolateVelocity<<<blocknum, threadnum>>>(buflist, pnum, fbuf);
	hipDeviceSynchronize();

	Kern_getmobu_walkthrough<<<blocknum, threadnum>>>(buflist, pnum);
	hipDeviceSynchronize();

	return;
}