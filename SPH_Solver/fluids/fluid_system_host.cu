#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#include <conio.h>


#include "fluid_system_host.cuh"		
#include "fluid_system_kern.cuh"
#include "MpmSolver.cuh"
#include "sph_solid.cuh"

#include "thrust\device_vector.h"	//thrust libs
#include "thrust\sort.h" 


FluidParams		fcuda;
bufList			fbuf;

hipError_t error;
extern ParamCarrier hostCarrier;

void cudaExit (int argc, char **argv)
{
	//CUT_EXIT(argc, argv); 
}
void cudaInit(int argc, char **argv)
{   
	//CUT_DEVICE_INIT(argc, argv);
	
	hipDeviceProp_t p;
	hipGetDeviceProperties ( &p, 0);
	
	printf ( "-- CUDA --\n" );
	printf ( "Name:       %s\n", p.name );
	printf ( "Revision:   %d.%d\n", p.major, p.minor );
	printf ( "Global Mem: %d\n", p.totalGlobalMem );
	printf ( "Shared/Blk: %d\n", p.sharedMemPerBlock );
	printf ( "Regs/Blk:   %d\n", p.regsPerBlock );
	printf ( "Warp Size:  %d\n", p.warpSize );
	printf ( "Mem Pitch:  %d\n", p.memPitch );
	printf ( "Thrds/Blk:  %d\n", p.maxThreadsPerBlock );
	printf ( "Const Mem:  %d\n", p.totalConstMem );
	printf ( "Clock Rate: %d\n", p.clockRate );	
};

int iDivUp (int totalnum, int threadnum) {
	
	if(threadnum==0)
		return 1;

	return (totalnum % threadnum != 0) ? (totalnum / threadnum + 1) : (totalnum / threadnum);
}

inline bool isPowerOfTwo(int n) { return ((n&(n-1))==0) ; }
inline int floorPow2(int n) {
	#ifdef WIN32
		return 1 << (int)logb((float)n);
	#else
		int exp;
		frexp((float)n, &exp);
		return 1 << (exp - 1);
	#endif
}

// Compute number of blocks to create
void computeNumBlocks (int numPnts, int maxThreads, int &numBlocks, int &numThreads)
{
	numThreads = min( maxThreads, numPnts );
	numBlocks = iDivUp ( numPnts, numThreads );
	if(numThreads==0)
		numThreads = 1;
}
#define CUDA_SAFE_CALL

void FluidClearCUDA ()
{
	hipFree(fbuf.displayBuffer);
	hipFree(fbuf.calcBuffer);

	hipFree ( fbuf.msortbuf );	
	hipFree(fbuf.MFidTable);
	//new sort
	hipFree(fbuf.mgcell);
	hipFree(fbuf.mgndx);
	hipFree(fbuf.mgridcnt);
	hipFree ( fbuf.midsort );
	hipFree ( fbuf.mgridoff );

}









void FluidSetupCUDA(ParamCarrier& params){
	fcuda.pnum = params.num;
	fcuda.gridTotal = params.gridtotal;
	
	// Compute number of blocks and threads
	computeNumBlocks(fcuda.pnum, 384, fcuda.numBlocks, fcuda.numThreads);			// particles
	computeNumBlocks(fcuda.gridTotal, 384, fcuda.gridBlocks, fcuda.gridThreads);		// grid cell

	// Allocate particle buffers
	//fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);
	fcuda.szPnts = params.maxNum;

	hipMalloc(&fbuf.displayBuffer, fcuda.szPnts * sizeof(displayPack));
	hipMalloc(&fbuf.calcBuffer,	fcuda.szPnts*sizeof(calculationPack));
	int temp_size = (sizeof(displayPack) + sizeof(calculationPack));
	hipMalloc(&fbuf.msortbuf, fcuda.szPnts*temp_size);

	//without index sort
	hipMalloc(&fbuf.densityResidue, fcuda.szPnts * sizeof(float));
	hipMalloc(&fbuf.press_l,  fcuda.szPnts*sizeof(float));
	hipMalloc(&fbuf.press_l1, fcuda.szPnts*sizeof(float));
	hipMalloc(&fbuf.rho_adv, fcuda.szPnts*sizeof(float));
	hipMalloc(&fbuf.stress,  fcuda.szPnts*sizeof(cmat3));
	hipMalloc(&fbuf.aii, fcuda.szPnts*sizeof(float));
	hipMalloc(&fbuf.dii, fcuda.szPnts*sizeof(cfloat3));
	hipMalloc(&fbuf.dijpj, fcuda.szPnts*sizeof(cfloat3));

	hipMalloc(&fbuf.MFidTable,	fcuda.szPnts*sizeof(int));
	

	// Allocate grid
	fcuda.szGrid = (fcuda.gridBlocks * fcuda.gridThreads);
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgcell, fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgndx,  fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgridcnt, fcuda.szGrid*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.midsort,  fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgridoff, fcuda.szGrid*sizeof(int)));
	

	//MpmAllocateBuffer();
}


void GetParticleIndexCUDA()
{
	hipMemset ( fbuf.mgridcnt, 0,			fcuda.gridTotal * sizeof(int));
	hipMemset ( fbuf.mgridoff, 0,			fcuda.gridTotal * sizeof(int));
	hipMemset ( fbuf.mgcell, 0,			fcuda.pnum * sizeof(uint));
	InitialSort<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: InsertParticlesCUDA: %s\n", hipGetErrorString(error) );
	}  
	hipDeviceSynchronize ();

}
void GetGridListCUDA()
{

	thrust::device_ptr<uint> dev_keysg(fbuf.mgcell);
	thrust::device_ptr<uint> dev_valuesg(fbuf.midsort);
	thrust::sort_by_key(dev_keysg,dev_keysg+fcuda.pnum,dev_valuesg);
	//hipDeviceSynchronize ();
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA ERROR: Thrust sort: %s\n", hipGetErrorString(error));
	}

	CalcFirstCnt <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	//	hipDeviceSynchronize ();
	hipDeviceSynchronize ();


	GetCnt <<<fcuda.numBlocks,fcuda.numThreads>>> (fbuf,fcuda.pnum);
	hipDeviceSynchronize ();
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA ERROR: Sort Grid: %s\n", hipGetErrorString(error));
	}
}

void RearrageDataCUDA()
{
	// Transfer particle data to temp buffers
	int n = fcuda.pnum;
	
	hipMemcpy ( fbuf.msortbuf + n*BUF_DISPLAYBUF, fbuf.displayBuffer, n*sizeof(displayPack), hipMemcpyDeviceToDevice);
	hipMemcpy(fbuf.msortbuf + n*BUF_CALCBUF, fbuf.calcBuffer, n*sizeof(calculationPack), hipMemcpyDeviceToDevice);
	//hipMemcpy(fbuf.msortbuf + n*BUF_INTMBUF, fbuf.intmBuffer, n*sizeof(IntermediatePack), hipMemcpyDeviceToDevice);

	// Counting Sort - pass one, determine grid counts
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR:CopyToSortBufferCUDA: %s\n", hipGetErrorString(error) );
	} 

	RearrangeData <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	hipDeviceSynchronize ();

	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR:Sorting Failed: %s\n", hipGetErrorString(error) );
	} 

}






void MpmAllocateBufferCUDA(ParamCarrier& param){

    //calculate size
    fcuda.mpmSize = param.mpmNodeNum;

    //int splitnum = 2;

    computeNumBlocks ( fcuda.mpmSize, 384, fcuda.mpmBlocks, fcuda.mpmThreads);

    hipMalloc(&fbuf.mpmMass,   fcuda.mpmSize * sizeof(float));
	hipMalloc(&fbuf.mpmPos, fcuda.mpmSize * sizeof(cfloat3));
	hipMalloc(&fbuf.mpmVel, fcuda.mpmSize * sizeof(cfloat3));
    //hipMalloc(&fbuf.mpmAlpha, fcuda.mpmSize * sizeof(float) * MAX_FLUIDNUM);
    //hipMalloc(&fbuf.mpmForce, fcuda.mpmSize * sizeof(cfloat3) * splitnum);
	hipMalloc(&fbuf.mpmGid,		fcuda.mpmSize * sizeof(uint));
}


void IndexMPMSortCUDA(){
    initMpm <<< fcuda.mpmBlocks, fcuda.mpmThreads >>> (fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();
}


void MpmColorTestCUDA() {
	MpmColorTest <<< fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
	hipDeviceSynchronize();
}

void MpmGetMomentumCUDA() {
	MpmGetMomentum <<< fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
	hipDeviceSynchronize();
}


void initSPH(float* restdensity,int* mftype){
	
    //initDensity<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	//hipDeviceSynchronize();

	/*CUDA_SAFE_CALL( hipMemcpy( restdensity, fbuf.mf_restdensity, fcuda.pnum*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( mftype,      fbuf.MFtype,         fcuda.pnum*sizeof(int),   hipMemcpyDeviceToHost));

	double sum=0;
	int cnt=0;
	for(int i=0; i<fcuda.pnum; i++){
		if( mftype[i]==1){
			sum += restdensity[i];
			cnt++;
		}
	}
	if (cnt > 0)
		sum /= cnt;
	CUDA_SAFE_CALL( hipMemcpy( fbuf.mf_restdensity,restdensity, fcuda.pnum*sizeof(float), hipMemcpyHostToDevice));
	printf("average density %f\n",sum);*/
    
    //MpmSortGridCuda();

    //initMpm<<<fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    //hipDeviceSynchronize();	
}


void MfComputePressureCUDA ()
{
	//ComputeBoundaryVolume <<< fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	//hipDeviceSynchronize();

	ComputeDensityPressure <<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();
	
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("%s\n", hipGetErrorString (error));
	}
}

void ComputeDensityIISPH_CUDA() {
	ComputeDensityIISPH <<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

//void MfComputeDriftVelCUDA ()
//{
//    //mfComputeDriftVel<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
//	
//    error = hipGetLastError();
//	if (error != hipSuccess) {
//		fprintf ( stderr, "CUDA ERROR: MfComputeDriftVelCUDA: %s\n", hipGetErrorString(error) );
//	}    
//	hipDeviceSynchronize ();
//}

//void MfComputeAlphaAdvanceCUDA ()
//{
//	//mfComputeAlphaAdvance<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
//	
//    error = hipGetLastError();
//	if (error != hipSuccess) {
//		fprintf ( stderr, "CUDA ERROR: MfComputeAlphaAdvanceCUDA: %s\n", hipGetErrorString(error) );
//	}    
//	hipDeviceSynchronize ();
//}
//void MfComputeCorrectionCUDA ()
//{
//	//mfComputeCorrection<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );	
//	
//	error = hipGetLastError();
//	if (error != hipSuccess) {
//		fprintf ( stderr, "CUDA ERROR: MfComputeCorrectionCUDA: %s\n", hipGetErrorString(error) );
//	}    
//	hipDeviceSynchronize ();
//}

void MfAdvanceCUDA ()
{
    AdvanceParticles<<< fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum );	
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfAdvanceCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void ComputeForceCUDA_ProjectU(){

	//pressure force, diffusion force
	//ComputeForce_projectu<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

	//ComputeSPHtensor<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputSPHtensor: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

	//AddSPHtensorForce<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	
	hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: Adding SPH tensor Force: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();
}

void MfComputeForceCUDA(){

    //SurfaceDetection <<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
    //hipDeviceSynchronize();

	ComputeForce<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

    //SurfaceTension<<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
    //error = hipGetLastError();
	//if (error != hipSuccess)
	//	fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
    //hipDeviceSynchronize ();
}



void ComputeMpmForce(){
    
    //Get Grid Mass and Velocity - 1
    //GetGridMassVel <<< fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();

    //Update Particle Strain Tensor - 2
    //CalcMpmParticleTensor <<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
    hipDeviceSynchronize();

    //Update Grid Force and Velocity - 3
    //CalcMpmGridForce<<<fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();

    //Update Particle Position and Velocity - 4
    //UpdateMpmParticlePos<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
    hipDeviceSynchronize();

}







//Newly updated 
void ComputeSolidTensorCUDA(){
	//velocity gradient, Strain, Stress
	ComputeSolidTensor<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void ComputeSolidForceCUDA(){
	ComputeSolidForce<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}


void ComputeSolidTensorX_CUDA() {
	//deformation gradient, Strain, Stress
	ComputeSolidTensor_X<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void ComputeSolidForceX_CUDA() {
	ComputeSolidForce_X<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void InitializeSolid_CUDA(){

	GetParticleIndexCUDA();
	GetGridListCUDA();
	RearrageDataCUDA();

	//calculate invA
	ComputeInvA <<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}


//IISPH
void ComputeBoundaryDensity() {
	ComputeBoundaryVolume <<< fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void PredictAdvection() {
	//v_adv, dii
	ComputeDii <<<fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();

	//rho_adv_i, aii
	ComputeAii <<<fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void PressureSolve() {
	int iter = 0;
	float rho_avg = 0;

	while (true) {
		Pressure_DP<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
		hipDeviceSynchronize();

		Pressure_Iter<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
		hipDeviceSynchronize();

		hipMemcpy(fbuf.press_l,fbuf.press_l1,sizeof(float)*fcuda.pnum,hipMemcpyDeviceToDevice);

		//criterion
		thrust::device_ptr<float> d_ptr = thrust::device_pointer_cast(fbuf.densityResidue);
		float sum = thrust::reduce(d_ptr, d_ptr+fcuda.pnum);
		sum /= fcuda.pnum;
		printf("%d iteration: residue %f\n",iter,sum);
		//break;

		if (sum<0.1 && iter>=2) {
			break;
		}
		iter++;
	}


}

void Integration() {
	IntegrateIISPH<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf,fcuda.pnum);
	hipDeviceSynchronize();
}