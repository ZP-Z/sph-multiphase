#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#include <conio.h>
//#include <cutil.h>					// cutil32.lib
//#include <cutil_math.h>				// cutil32.lib

//#include <GL/glut.h>
//#include <cuda_gl_interop.h>

#include "fluid_system_host.cuh"		
#include "fluid_system_kern.cuh"
//#include "radixsort.cu"						// Build in RadixSort
#include "thrust\device_vector.h"	//thrust libs
#include "thrust\sort.h" 


FluidParams		fcuda;
//extern ParamCarrier hostCarrier;
bufList			fbuf;
hipError_t error;

__device__ uint			gridActive;
__device__ int			flagNumFT;  //for transfer
__device__ int			pNumFT;		//for transfer

#define BLOCK_SIZE 256
#define LOCAL_PMAX		896
#define NUM_CELL		27
#define LAST_CELL		26
#define CENTER_CELL		13

float**			g_scanBlockSums;
int**			g_scanBlockSumsInt;
unsigned int	g_numEltsAllocated = 0;
unsigned int	g_numLevelsAllocated = 0;

void cudaExit (int argc, char **argv)
{
	//CUT_EXIT(argc, argv); 
}
void cudaInit(int argc, char **argv)
{   
	//CUT_DEVICE_INIT(argc, argv);
	
	hipDeviceProp_t p;
	hipGetDeviceProperties ( &p, 0);
	
	printf ( "-- CUDA --\n" );
	printf ( "Name:       %s\n", p.name );
	printf ( "Revision:   %d.%d\n", p.major, p.minor );
	printf ( "Global Mem: %d\n", p.totalGlobalMem );
	printf ( "Shared/Blk: %d\n", p.sharedMemPerBlock );
	printf ( "Regs/Blk:   %d\n", p.regsPerBlock );
	printf ( "Warp Size:  %d\n", p.warpSize );
	printf ( "Mem Pitch:  %d\n", p.memPitch );
	printf ( "Thrds/Blk:  %d\n", p.maxThreadsPerBlock );
	printf ( "Const Mem:  %d\n", p.totalConstMem );
	printf ( "Clock Rate: %d\n", p.clockRate );	
};

int iDivUp (int totalnum, int threadnum) {
	
	if(threadnum==0)
		return 1;

	return (totalnum % threadnum != 0) ? (totalnum / threadnum + 1) : (totalnum / threadnum);
}

inline bool isPowerOfTwo(int n) { return ((n&(n-1))==0) ; }
inline int floorPow2(int n) {
	#ifdef WIN32
		return 1 << (int)logb((float)n);
	#else
		int exp;
		frexp((float)n, &exp);
		return 1 << (exp - 1);
	#endif
}

// Compute number of blocks to create
void computeNumBlocks (int numPnts, int maxThreads, int &numBlocks, int &numThreads)
{
	numThreads = min( maxThreads, numPnts );
	numBlocks = iDivUp ( numPnts, numThreads );
	if(numThreads==0)
		numThreads = 1;
}
#define CUDA_SAFE_CALL

void FluidClearCUDA ()
{
	CUDA_SAFE_CALL ( hipFree ( fbuf.mpos ) );	
	CUDA_SAFE_CALL ( hipFree ( fbuf.mvel ) );	
	CUDA_SAFE_CALL ( hipFree ( fbuf.mveleval ) );	
	CUDA_SAFE_CALL(hipFree(fbuf.mforce));
	CUDA_SAFE_CALL(hipFree(fbuf.mpress));
	CUDA_SAFE_CALL(hipFree(fbuf.last_mpress));
	CUDA_SAFE_CALL ( hipFree ( fbuf.mdensity ) );		
	CUDA_SAFE_CALL ( hipFree ( fbuf.mgcell ) );	
	CUDA_SAFE_CALL ( hipFree ( fbuf.mgndx ) );	
	CUDA_SAFE_CALL ( hipFree ( fbuf.mclr ) );	
	CUDA_SAFE_CALL ( hipFree ( fbuf.misbound ) );	

	//CUDA_SAFE_CALL ( hipFree ( fbuf.mcluster ) );	

	//multi fluid
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_alpha ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_alpha_pre ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_pressure_modify ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_vel_phrel) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_restdensity ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_restmass ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_visc ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_velxcor ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mf_alphagrad ) );

	CUDA_SAFE_CALL ( hipFree ( fbuf.msortbuf ) );	

	CUDA_SAFE_CALL ( hipFree ( fbuf.mgrid ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mgridcnt ) );
	//new sort
	CUDA_SAFE_CALL ( hipFree ( fbuf.midsort ) );

	CUDA_SAFE_CALL ( hipFree ( fbuf.mgridoff ) );
	CUDA_SAFE_CALL ( hipFree ( fbuf.mgridactive ) );
}

void FluidSetupCUDA(ParamCarrier& params){
	fcuda.pnum = params.num;
	fcuda.gridRes = params.gridres;
	fcuda.gridSize = params.gridsize;
	fcuda.gridDelta = params.gridIdfac;
	fcuda.gridMin = params.gridmin;
	fcuda.gridMax = params.gridmax;
	fcuda.gridTotal = params.gridtotal;
	fcuda.gridSrch = params.searchnum;
	fcuda.gridAdjCnt = params.neighbornum;
	fcuda.gridScanMax.x = params.gridres.x - params.searchnum;
	fcuda.gridScanMax.y = params.gridres.y - params.searchnum;
	fcuda.gridScanMax.z = params.gridres.z - params.searchnum;
	//fcuda.chk = chk;
	//fcuda.mf_up=0;

	// Build Adjacency Lookup
	int cell = 0;
	for (int y=0; y < params.searchnum; y++)
		for (int z=0; z < params.searchnum; z++)
			for (int x=0; x < params.searchnum; x++)
				fcuda.gridAdj[cell++]  = (y * fcuda.gridRes.z+ z)*fcuda.gridRes.x +  x;

	/*printf ( "CUDA Adjacency Table\n");
	for (int n=0; n < fcuda.gridAdjCnt; n++ ) {
	printf ( "  ADJ: %d, %d\n", n, fcuda.gridAdj[n] );
	}	*/

	// Compute number of blocks and threads
	computeNumBlocks(fcuda.pnum, 384, fcuda.numBlocks, fcuda.numThreads);			// particles
	computeNumBlocks(fcuda.gridTotal, 384, fcuda.gridBlocks, fcuda.gridThreads);		// grid cell
																						// Allocate particle buffers
	fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);
	/*printf ( "CUDA Allocate: \n" );
	printf ( "  Pnts: %d, t:%dx%d=%d, Size:%d\n", fcuda.pnum, fcuda.numBlocks, fcuda.numThreads, fcuda.numBlocks*fcuda.numThreads, fcuda.szPnts);
	printf ( "  Grid: %d, t:%dx%d=%d, bufGrid:%d, Res: %dx%dx%d\n", fcuda.gridTotal, fcuda.gridBlocks, fcuda.gridThreads, fcuda.gridBlocks*fcuda.gridThreads, fcuda.szGrid, (int) fcuda.gridRes.x, (int) fcuda.gridRes.y, (int) fcuda.gridRes.z );
	*/

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mpos, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mvel, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mveleval, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mforce, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)* 3));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mpress, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	//CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.last_mpress, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mdensity, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgcell, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgndx, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mclr, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)));
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mColor,    EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float4)));

	int temp_size = EMIT_BUF_RATIO*(4*(sizeof(float)*3) + 2*sizeof(float) + 3*sizeof(uint));

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.misbound, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int)));
	temp_size += EMIT_BUF_RATIO*sizeof(int);
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.accel, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(cfloat3)));
	temp_size += EMIT_BUF_RATIO*sizeof(cfloat3);

	//multi fluid
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_alpha, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*MAX_FLUIDNUM));    //float* num
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_alpha_pre, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*MAX_FLUIDNUM));    //float* num

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_vel_phrel, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3*MAX_FLUIDNUM));	//float*3*num
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_alphagrad, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3*MAX_FLUIDNUM));   //float*3*num

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_pressure_modify, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));				//float
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_restmass, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_restdensity, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_visc, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_velxcor, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3));

	temp_size += EMIT_BUF_RATIO*(2*MAX_FLUIDNUM*sizeof(float) + 2*MAX_FLUIDNUM*(sizeof(float)*3) + 4*sizeof(float) + sizeof(float)*3);

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFtype, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int))); //indicator function
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFid, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int))); //born id
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFidTable, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int))); //id table no sorting

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFtensor, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9)); //deformable tensor
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFRtensor, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFtemptensor, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.MFvelgrad, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9)); //no sorting
	CUDA_SAFE_CALL(hipMalloc(&fbuf.MFpepsilon, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float))); //no sorting

	temp_size += EMIT_BUF_RATIO*(sizeof(int)*2 + sizeof(float)*9*3);

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mf_multiFlag, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)));

	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.msortbuf, EMIT_BUF_RATIO*fcuda.szPnts*temp_size));

	// Allocate grid
	fcuda.szGrid = (fcuda.gridBlocks * fcuda.gridThreads);
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgrid, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgridcnt, fcuda.szGrid*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.midsort, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgridoff, fcuda.szGrid*sizeof(int)));
	CUDA_SAFE_CALL(hipMalloc((void**)&fbuf.mgridactive, fcuda.szGrid*sizeof(int)));

	//MpmAllocateBuffer();

	updateParam(&fcuda);
}

void FluidSetupCUDA ( int num, int gsrch, int3 res, cfloat3 size, cfloat3 delta, cfloat3 gmin, cfloat3 gmax, int total, int chk)
{	
	//fcuda.pnum = num;	
	//fcuda.gridRes = res;
	//fcuda.gridSize = size;
	//fcuda.gridDelta = delta;
	//fcuda.gridMin = gmin;
	//fcuda.gridMax = gmax;
	//fcuda.gridTotal = total;
	//fcuda.gridSrch = gsrch;
	//fcuda.gridAdjCnt = gsrch*gsrch*gsrch;
	////fcuda.gridScanMax = res;
	////fcuda.gridScanMax -= make_int3( fcuda.gridSrch, fcuda.gridSrch, fcuda.gridSrch );
	//fcuda.chk = chk;
	//fcuda.mf_up=0;

	//// Build Adjacency Lookup
	//int cell = 0;
	//for (int y=0; y < gsrch; y++ ) 
	//	for (int z=0; z < gsrch; z++ ) 
	//		for (int x=0; x < gsrch; x++ ) 
	//			fcuda.gridAdj [ cell++]  = ( y * fcuda.gridRes.z+ z )*fcuda.gridRes.x +  x ;			
	//
	///*printf ( "CUDA Adjacency Table\n");
	//for (int n=0; n < fcuda.gridAdjCnt; n++ ) {
	//	printf ( "  ADJ: %d, %d\n", n, fcuda.gridAdj[n] );
	//}	*/

	//// Compute number of blocks and threads
	//computeNumBlocks ( fcuda.pnum, 384, fcuda.numBlocks, fcuda.numThreads);			// particles
	//computeNumBlocks ( fcuda.gridTotal, 384, fcuda.gridBlocks, fcuda.gridThreads);		// grid cell
	//// Allocate particle buffers
	//fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);     
	///*printf ( "CUDA Allocate: \n" );
	//printf ( "  Pnts: %d, t:%dx%d=%d, Size:%d\n", fcuda.pnum, fcuda.numBlocks, fcuda.numThreads, fcuda.numBlocks*fcuda.numThreads, fcuda.szPnts);
	//printf ( "  Grid: %d, t:%dx%d=%d, bufGrid:%d, Res: %dx%dx%d\n", fcuda.gridTotal, fcuda.gridBlocks, fcuda.gridThreads, fcuda.gridBlocks*fcuda.gridThreads, fcuda.szGrid, (int) fcuda.gridRes.x, (int) fcuda.gridRes.y, (int) fcuda.gridRes.z );		
	//*/

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mpos,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3 ) );	
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mvel,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3 ) );	
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mveleval,	EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3 ) );	
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mforce,    EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)* 3));
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mpress,    EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.last_mpress, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)));
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mdensity,	EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float) ) );	
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgcell,	EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgndx,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint)) );	
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mclr,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint) ) );	
	////CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mColor,    EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float4)));
	//
	//int temp_size = EMIT_BUF_RATIO*(4*(sizeof(float)*3) + 3*sizeof(float) + 2*sizeof(int) + sizeof(uint));

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.misbound, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int)) );	
	//temp_size += EMIT_BUF_RATIO*sizeof(int);
	//CUDA_SAFE_CALL ( hipMalloc ((void**)&fbuf.accel, EMIT_BUF_RATIO*fcuda.szPnts*sizeof(cfloat3)));
	//temp_size += EMIT_BUF_RATIO*sizeof(cfloat3);

	////multi fluid
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_alpha,					EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*MAX_FLUIDNUM ) );    //float* num
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_alpha_pre,				EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*MAX_FLUIDNUM ) );    //float* num

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_vel_phrel,				EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3*MAX_FLUIDNUM ) );	//float*3*num
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_alphagrad,				EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3*MAX_FLUIDNUM ) );   //float*3*num

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_pressure_modify,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float) ) );				//float
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_restmass,				EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_restdensity,			EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_visc,					EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_velxcor,				EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*3 ) );

	//temp_size += EMIT_BUF_RATIO*(2*MAX_FLUIDNUM*sizeof(float) + 2*MAX_FLUIDNUM*(sizeof(float)*3) + 4*sizeof(float) + sizeof(float)*3 );

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFtype,					EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int) ) ); //indicator function
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFid,						EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int) ) ); //born id
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFidTable,					EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int) ) ); //id table no sorting

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFtensor,			EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9 ) ); //deformable tensor
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFRtensor,			EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9)  );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFtemptensor,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.MFvelgrad,			EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)*9) ); //no sorting
	//CUDA_SAFE_CALL ( hipMalloc ( &fbuf.MFpepsilon,					EMIT_BUF_RATIO*fcuda.szPnts*sizeof(float)) ); //no sorting

	//temp_size += EMIT_BUF_RATIO*(sizeof(int)*2 + sizeof(float)*9*3 );

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mf_multiFlag,	EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint) ) );

	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.msortbuf,	EMIT_BUF_RATIO*fcuda.szPnts*temp_size ) );	

	//// Allocate grid
	//fcuda.szGrid = (fcuda.gridBlocks * fcuda.gridThreads);  
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgrid,		EMIT_BUF_RATIO*fcuda.szPnts*sizeof(int) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgridcnt,	fcuda.szGrid*sizeof(int) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.midsort,	EMIT_BUF_RATIO*fcuda.szPnts*sizeof(uint) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgridoff,	fcuda.szGrid*sizeof(int) ) );
	//CUDA_SAFE_CALL ( hipMalloc ( (void**) &fbuf.mgridactive, fcuda.szGrid*sizeof(int) ) );
	//
 //   //MpmAllocateBuffer();

 //   updateParam( &fcuda );
}

void FluidParamCUDA ( float ss, float sr, float pr, float mass, float rest, cfloat3 bmin, cfloat3 bmax, float estiff, float istiff,float pbstiff, float visc, float damp, float fmin, float fmax, float ffreq, float gslope, float gx, float gy, float gz, float al, float vl )
{
	fcuda.psimscale = ss;
	fcuda.psmoothradius = sr;
	fcuda.pradius = pr;
	fcuda.r2 = sr * sr;
	fcuda.pmass = mass;
	fcuda.prest_dens = rest;	
	fcuda.pboundmin = bmin;
	fcuda.pboundmax = bmax;
	fcuda.pextstiff = estiff;
	fcuda.pintstiff = istiff;
	fcuda.pbstiff = pbstiff;
	fcuda.pvisc = visc;
	fcuda.pdamp = damp;
	fcuda.pforce_min = fmin;
	fcuda.pforce_max = fmax;
	fcuda.pforce_freq = ffreq;
	fcuda.pground_slope = gslope;
	fcuda.pgravity = cfloat3( gx, gy, gz );
	fcuda.AL = al;
	fcuda.AL2 = al * al;
	fcuda.VL = vl;
	fcuda.VL2 = vl * vl;

	printf ( "Bound Min: %f %f %f\n", bmin.x, bmin.y, bmin.z );
	printf ( "Bound Max: %f %f %f\n", bmax.x, bmax.y, bmax.z );

	fcuda.pdist = pow ( fcuda.pmass / fcuda.prest_dens, 1/3.0f );
	fcuda.poly6kern = 315.0f / (64.0f * 3.141592 * pow( sr, 9.0f) );
	fcuda.spikykern = -45.0f / (3.141592 * pow( sr, 6.0f) );
	fcuda.spikykernel = 15 / (3.141592 * pow(sr,6.0f));
	fcuda.lapkern = 45.0f / (3.141592 * pow( sr, 6.0f) );	

	updateParam( &fcuda );
	hipDeviceSynchronize ();
}

void FluidParamCUDA (ParamCarrier& params){
	fcuda.psimscale = params.simscale;
	fcuda.psmoothradius = params.smoothradius; //real smooth radius
	fcuda.pradius = params.radius;
	fcuda.r2 = params.smoothradius * params.smoothradius;
	fcuda.pmass = params.mass;
	fcuda.prest_dens = params.restdensity;
	fcuda.pvisc = params.viscosity;
	fcuda.pboundmin = params.softminx;
	fcuda.pboundmax = params.softmaxx;
	fcuda.pextstiff = params.extstiff;
	fcuda.pintstiff = params.intstiff;
//	fcuda.pbstiff = pbstiff;
	fcuda.pdamp = params.extdamp;
	//fcuda.pforce_min = fmin;
	//fcuda.pforce_max = fmax;
	//fcuda.pforce_freq = ffreq;
	//fcuda.pground_slope = gslope;
	fcuda.pgravity = params.gravity;
	fcuda.AL = params.acclimit;
	fcuda.AL2 = params.acclimit * params.acclimit;
	fcuda.VL = params.vlimit;
	fcuda.VL2 = params.vlimit * params.vlimit;

	printf("Bound Min: %f %f %f\n", fcuda.pboundmin.x, fcuda.pboundmin.y, fcuda.pboundmin.z);
	printf("Bound Max: %f %f %f\n", fcuda.pboundmax.x, fcuda.pboundmax.y, fcuda.pboundmax.z);

	fcuda.pdist = pow(fcuda.pmass / fcuda.prest_dens, 1/3.0f);
	fcuda.poly6kern = 315.0f / (64.0f * 3.141592 * pow(fcuda.psmoothradius, 9.0f));
	fcuda.spikykern = -45.0f / (3.141592 * pow(fcuda.psmoothradius, 6.0f));
	fcuda.spikykernel = 15 / (3.141592 * pow(fcuda.psmoothradius, 6.0f));
	fcuda.lapkern = 45.0f / (3.141592 * pow(fcuda.psmoothradius, 6.0f));

	//fcuda.mf_catnum = catnum;
	//fcuda.mf_diffusion = diffusion;
	fcuda.mf_dt = params.dt;
	/*for (int i=0; i<MAX_FLUIDNUM; i++)
	{
		fcuda.mf_dens[i] = dens[i];
		fcuda.mf_visc[i] = visc[i];
	}*/

	updateParam(&fcuda);
	hipDeviceSynchronize();
}

void FluidParamCUDAbuffer_projectu(float* buffer){
	fcuda.coK =					buffer[0];//coK;
	fcuda.coG =					buffer[1];//coG;
	fcuda.phi =					buffer[2];//phi;
	fcuda.coA =					buffer[3];//coA;
	fcuda.coB =					buffer[4];//coB;
	fcuda.coLambdaK =			buffer[5];//coLambdaK;
	fcuda.cohesion =			buffer[6];//cohesion;
	fcuda.boundaryVisc =  		buffer[7];//boundaryVisc;
	fcuda.sleepvel =  			buffer[8];//sleepvel;
	fcuda.initspacing =			buffer[9];//initspacing;
	
	fcuda.coN =					buffer[10];//coN;
	fcuda.Yradius =  			buffer[11];//Yradius;
	fcuda.visc_factor = 		buffer[12];// visc_factor;
	fcuda.fluid_pfactor = 		buffer[13];// fluid_pfactor;
	fcuda.solid_pfactor =  		buffer[14];//solid_pfactor;
	fcuda.fsa = 				buffer[15];// fsa;
	fcuda.fsb =  				buffer[16];//fsb;
	fcuda.bdamp = 				buffer[17];// bdamp;
	fcuda.coD =  				buffer[18];//coD;
	fcuda.coD0 = 				buffer[19];//coD0;
	
	fcuda.solid_coG =  			buffer[20];//solid_coG;
//	fcuda.solid_coV =  			buffer[21];//solid_coV;
	fcuda.solid_coK =  			buffer[22];//solid_coK;
	fcuda.solid_coA =  			buffer[23];//solid_coA;
	fcuda.solid_coB =  			buffer[24];//solid_coB;
	fcuda.solid_fsa =  			buffer[25];//solid_fsa;
	fcuda.solid_fsb =  			buffer[26];//solid_fsb;
	fcuda.solid_coN =  			buffer[27];//solid_coN;
	fcuda.solid_phi =  			buffer[28];//solid_phi;
	fcuda.solid_Yradius =  		buffer[29];//solid_Yradius;
	fcuda.fluidVConstraint =  	buffer[30];//fluidVConstraint;
	fcuda.tohydro =  			buffer[31];//tohydro;

    fcuda.mpmSpacing = buffer[32]; //mpmSpacing
    fcuda.minVec = cfloat3(buffer[33], buffer[34], buffer[35]);
    fcuda.maxVec = cfloat3(buffer[36], buffer[37], buffer[38]);
    
}

void FluidMfParamCUDA ( float *dens, float *visc, float diffusion, float catnum, float dt,  cfloat3 cont, cfloat3 mb1,cfloat3 mb2, float relax,int example)
{
	fcuda.mf_catnum = catnum;
	fcuda.mf_diffusion = diffusion;
	fcuda.mf_dt = dt;
	for(int i=0;i<MAX_FLUIDNUM;i++)
	{
		fcuda.mf_dens[i] = dens[i];
		fcuda.mf_visc[i] = visc[i];
	}
	fcuda.mf_multiFlagPNum = 0;
	//fcuda.mf_splitVolume = splitV;
	//fcuda.mf_mergeVolume = mergeV;
	fcuda.mf_maxPnum = fcuda.pnum * EMIT_BUF_RATIO;
	fcuda.cont =  cont.x;	fcuda.cont1 = cont.y;	fcuda.cont2 = cont.z;	
	fcuda.mb1.x = mb1.x;	fcuda.mb1.y = mb1.y;	fcuda.mb1.z = mb1.z;
	fcuda.mb2.x = mb2.x;	fcuda.mb2.y = mb2.y;	fcuda.mb2.z = mb2.z;
	fcuda.bxmin = mb1.x;    fcuda.by = mb1.y;       fcuda.bzmin = mb1.z;
	fcuda.bxmax = mb2.x;							fcuda.bzmax = mb2.z; 
	
	fcuda.relax = relax;
	fcuda.example = example;
	updateParam( &fcuda );
	hipDeviceSynchronize ();
}


//Copy buffers
void CopyToCUDA ( float* pos, float* vel, float* veleval, float* force, float* pressure, float* density, uint* cluster, uint* gnext, char* clr)
{

	// Send particle buffers
	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mpos,		pos,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ) );	
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mvel,		vel,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mveleval, veleval,		numPoints*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mforce,	force,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mpress,	pressure,		numPoints*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mdensity, density,		numPoints*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mclr,		clr,			numPoints*sizeof(uint), hipMemcpyHostToDevice ) );


	hipDeviceSynchronize ();	
}
void CopyMfToCUDA ( float* alpha, float* alpha_pre, float* pressure_modify, float* vel_phrel, float* restmass, float* restdensity, float* visc, float* velxcor, float* alphagrad)
{
	// Send particle buffers
	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alpha,				alpha,				numPoints*MAX_FLUIDNUM*sizeof(float), hipMemcpyHostToDevice ) );	
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alpha_pre,			alpha_pre,			numPoints*MAX_FLUIDNUM*sizeof(float), hipMemcpyHostToDevice ) );

	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_vel_phrel,			vel_phrel,			numPoints*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alphagrad,			alphagrad,			numPoints*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyHostToDevice ) );

	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_pressure_modify,	pressure_modify,	numPoints*sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_restmass,			restmass,			numPoints*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_restdensity,		restdensity,		numPoints*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_visc,				visc,				numPoints*sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_velxcor,			velxcor,			numPoints*sizeof(float)*3, hipMemcpyHostToDevice ) );
	//CUDA_SAFE_CALL( hipMemcpy ( fbuf.MFtype,				mftype,				numPoints*sizeof(int), hipMemcpyHostToDevice ) );
	hipDeviceSynchronize ();	
}
void CopyBoundToCUDA (int* isbound )
{
	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.misbound,	isbound,		numPoints*sizeof(int), hipMemcpyHostToDevice ) );
	hipDeviceSynchronize ();	
}
void CopyToCUDA_Uproject(int* mftype, float* tensorbuffer, int* bornid){
	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtype, mftype, numPoints*sizeof(int), hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtensor, tensorbuffer, numPoints*sizeof(float)*9, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtemptensor, tensorbuffer, numPoints*sizeof(float)*9, hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFid, bornid, numPoints*sizeof(int), hipMemcpyHostToDevice));
	hipDeviceSynchronize ();
}

void CopyFromCUDA ( float* pos, float* vel, float* veleval, float* force, float* pressure, float* density, uint* cluster, uint* gnext, char* clr, int mode)
{
	// Return particle buffers
	int numPoints = fcuda.pnum;

	CUDA_SAFE_CALL( hipMemcpy ( pos,		fbuf.mpos,			numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
	CUDA_SAFE_CALL( hipMemcpy ( clr,		fbuf.mclr,			numPoints*sizeof(uint),  hipMemcpyDeviceToHost ) );
	CUDA_SAFE_CALL(hipMemcpy(pressure, fbuf.mpress, numPoints*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL(hipMemcpy(veleval, fbuf.mforce, numPoints*sizeof(float)*3, hipMemcpyDeviceToHost));

	if( mode == 2){
		CUDA_SAFE_CALL( hipMemcpy ( vel,		fbuf.mvel,			numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( veleval,	fbuf.mveleval,		numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( force,		fbuf.mforce,		numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
		
		CUDA_SAFE_CALL( hipMemcpy ( density,	fbuf.mdensity,		numPoints*sizeof(float),  hipMemcpyDeviceToHost ) );
	}
	
	hipDeviceSynchronize ();	
}
void CopyMfFromCUDA ( float* alpha, float* alpha_pre, float* pressure_modify, float* vel_phrel, float* restmass, float* restdensity, float* visc, float* velxcor, float* alphagrad, int mode)
{

	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy ( alpha,				fbuf.mf_alpha,				numPoints*MAX_FLUIDNUM*sizeof(float), hipMemcpyDeviceToHost ) );
	CUDA_SAFE_CALL( hipMemcpy ( restmass,			fbuf.mf_restmass,			numPoints*sizeof(float),  hipMemcpyDeviceToHost ) );
	CUDA_SAFE_CALL( hipMemcpy ( restdensity,		fbuf.mf_restdensity,		numPoints*sizeof(float),  hipMemcpyDeviceToHost ) );
	if( mode == 2){
		// Send particle buffers
		
		CUDA_SAFE_CALL( hipMemcpy ( alpha_pre,			fbuf.mf_alpha_pre,			numPoints*MAX_FLUIDNUM*sizeof(float), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( pressure_modify,	fbuf.mf_pressure_modify,	numPoints*sizeof(float), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( vel_phrel,			fbuf.mf_vel_phrel,			numPoints*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyDeviceToHost ) );

		CUDA_SAFE_CALL( hipMemcpy ( visc,				fbuf.mf_visc,				numPoints*sizeof(float), hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( velxcor,			fbuf.mf_velxcor,			numPoints*sizeof(float)*3, hipMemcpyDeviceToHost ) );
		CUDA_SAFE_CALL( hipMemcpy ( alphagrad,			fbuf.mf_alphagrad,			numPoints*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyDeviceToHost ) );
	}
}
void CopyBoundFromCUDA (int* isbound )
{
	int numPoints = fcuda.pnum;
	if ( isbound != 0x0 ) CUDA_SAFE_CALL( hipMemcpy ( isbound,	fbuf.misbound,		numPoints*sizeof(int),  hipMemcpyDeviceToHost ) );
	hipDeviceSynchronize ();	
}
void CopyFromCUDA_Uproject(int* mftype, int* idtable, float* pepsilon, float* stensor, int mode){
	int numPoints = fcuda.pnum;
	CUDA_SAFE_CALL( hipMemcpy( mftype, fbuf.MFtype, numPoints*sizeof(int), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( idtable, fbuf.MFid, numPoints*sizeof(int), hipMemcpyDeviceToHost));
	//CUDA_SAFE_CALL( hipMemcpy( pepsilon, fbuf.MFpepsilon, numPoints*sizeof(float), hipMemcpyDeviceToHost));
	
	if( mode==2) //to save particles
		CUDA_SAFE_CALL( hipMemcpy( stensor, fbuf.MFtemptensor, numPoints*sizeof(float)*9, hipMemcpyDeviceToHost));
	hipDeviceSynchronize ();
}


//Called when particles emitted
void CopyEmitToCUDA ( float* pos, float* vel, float* veleval, float* force, float* pressure, float* density, uint* cluster, uint* gnext, char* clr, int startnum, int numcount,int* isbound )
{

	// Send particle buffers
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mpos+startnum,		pos+startnum*3,			numcount*sizeof(float)*3, hipMemcpyHostToDevice ) );	
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mvel+startnum,		vel+startnum*3,			numcount*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mveleval+startnum,	veleval+startnum*3,		numcount*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mforce+startnum,	force+startnum*3,			numcount*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mpress+startnum,		pressure+startnum,		numcount*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mdensity+startnum,	density+startnum,		numcount*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mclr+startnum,		clr+startnum,			numcount*sizeof(uint), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.misbound + startnum,				isbound + startnum,				numcount*sizeof(int), hipMemcpyHostToDevice ) );

	hipDeviceSynchronize ();	
}	
void CopyEmitMfToCUDA ( float* alpha, float* alpha_pre, float* pressure_modify, float* vel_phrel, float* restmass, float* restdensity, float* visc, float* velxcor, float* alphagrad,int startnum, int numcount)
{
	// Send particle buffers
	int mulstartnum = startnum*MAX_FLUIDNUM;
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alpha + mulstartnum,				alpha + mulstartnum,				numcount*MAX_FLUIDNUM*sizeof(float), hipMemcpyHostToDevice ) );	
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alpha_pre + mulstartnum,			alpha_pre + mulstartnum,			numcount*MAX_FLUIDNUM*sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_pressure_modify+startnum,			pressure_modify+startnum,			numcount*sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_vel_phrel + mulstartnum,			vel_phrel + mulstartnum*3,			numcount*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_restmass+startnum,					restmass+startnum,					numcount*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_restdensity+startnum,				restdensity+startnum,				numcount*sizeof(float),  hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_visc+startnum,						visc+startnum,						numcount*sizeof(float), hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_velxcor+startnum,					velxcor+startnum*3,					numcount*sizeof(float)*3, hipMemcpyHostToDevice ) );
	CUDA_SAFE_CALL( hipMemcpy ( fbuf.mf_alphagrad + mulstartnum,			alphagrad + mulstartnum*3,			numcount*MAX_FLUIDNUM*sizeof(float)*3, hipMemcpyHostToDevice ) );

	hipDeviceSynchronize ();	
}

void CopyEmitToCUDA_Uproject(int* mftype, float* tensorbuffer, int* bornid, int startnum, int numcount){
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtype + startnum,			mftype+startnum,				numcount*sizeof(int),			hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtensor + startnum*9,		tensorbuffer+startnum*9,		numcount*sizeof(float)*9,		hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFtemptensor + startnum*9, tensorbuffer+startnum*9,		numcount*sizeof(float)*9,		hipMemcpyHostToDevice));
	CUDA_SAFE_CALL( hipMemcpy( fbuf.MFid + startnum,			bornid+startnum,				numcount*sizeof(int),			hipMemcpyHostToDevice));
	
	hipDeviceSynchronize ();
}

void UpdatePNumCUDA( int newPnum)
{
	fcuda.pnum = newPnum;
	computeNumBlocks ( fcuda.pnum, 384, fcuda.numBlocks, fcuda.numThreads);    //threads changed!
	fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);					   //szPnts changed!	
	updateParam( &fcuda );
	hipDeviceSynchronize ();
}
int MfGetPnum(){
	return fcuda.pnum;
}


//Called in RunSimulateCudaFull
void InitialSortCUDA( uint* gcell, uint* ccell, int* gcnt )
{
	hipMemset ( fbuf.mgridcnt, 0,			fcuda.gridTotal * sizeof(int));
	hipMemset ( fbuf.mgridoff, 0,			fcuda.gridTotal * sizeof(int));
	hipMemset ( fbuf.mgcell, 0,			fcuda.pnum * sizeof(uint));
	InitialSort<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr,  "CUDA ERROR: InsertParticlesCUDA: %s\n", hipGetErrorString(error) );
	}  
	hipDeviceSynchronize ();

	// Transfer data back if requested (for validation)
	if (gcell != 0x0) {
		CUDA_SAFE_CALL( hipMemcpy ( gcell,	fbuf.mgcell,	fcuda.pnum*sizeof(uint),		hipMemcpyDeviceToHost ) );		
		CUDA_SAFE_CALL( hipMemcpy ( gcnt,	fbuf.mgridcnt,	fcuda.gridTotal*sizeof(int),	hipMemcpyDeviceToHost ) );
	}
}
void SortGridCUDA( int* goff )
{

	thrust::device_ptr<uint> dev_keysg(fbuf.mgcell);
	thrust::device_ptr<uint> dev_valuesg(fbuf.midsort);
	thrust::sort_by_key(dev_keysg,dev_keysg+fcuda.pnum,dev_valuesg);
	//hipDeviceSynchronize ();
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA ERROR: Thrust sort: %s\n", hipGetErrorString(error));
	}

	CalcFirstCnt <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	//	hipDeviceSynchronize ();
	hipDeviceSynchronize ();


	GetCnt <<<fcuda.numBlocks,fcuda.numThreads>>> (fbuf,fcuda.pnum);
	hipDeviceSynchronize ();
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf(stderr, "CUDA ERROR: Sort Grid: %s\n", hipGetErrorString(error));
	}
}

void CountingSortFullCUDA_( uint* ggrid )
{
	// Transfer particle data to temp buffers
	int n = fcuda.pnum;
	hipMemcpy ( fbuf.msortbuf + n*BUF_POS,		fbuf.mpos,		n*sizeof(float)*3,	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_VEL,		fbuf.mvel,		n*sizeof(float)*3,	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_VELEVAL,	fbuf.mveleval,	n*sizeof(float)*3,	hipMemcpyDeviceToDevice );
	hipMemcpy(fbuf.msortbuf + n*BUF_FORCE,		fbuf.mforce, n*sizeof(float)* 3, hipMemcpyDeviceToDevice);
	hipMemcpy(fbuf.msortbuf + n*BUF_PRESS,		fbuf.mpress, n*sizeof(float), hipMemcpyDeviceToDevice);
	//hipMemcpy(fbuf.msortbuf + n*BUF_LAST_PRESS, fbuf.last_mpress, n*sizeof(float), hipMemcpyDeviceToDevice);
	hipMemcpy ( fbuf.msortbuf + n*BUF_DENS,	fbuf.mdensity,	n*sizeof(float),	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_GCELL,	fbuf.mgcell,	n*sizeof(uint),		hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_GNDX,	fbuf.mgndx,		n*sizeof(uint),		hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_CLR,		fbuf.mclr,		n*sizeof(uint),		hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_ISBOUND,		fbuf.misbound,		n*sizeof(int),		hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_ACCEL,	fbuf.accel,		n*sizeof(cfloat3),	hipMemcpyDeviceToDevice);

	//multi fluid
	hipMemcpy ( fbuf.msortbuf + n*BUF_ALPHA,			fbuf.mf_alpha,			n*MAX_FLUIDNUM*sizeof(float),	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_ALPHAPRE,		fbuf.mf_alpha_pre,		n*MAX_FLUIDNUM*sizeof(float),	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_PRESSMODI,		fbuf.mf_pressure_modify,n*sizeof(float),				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_VELPHREL,		fbuf.mf_vel_phrel,		n*MAX_FLUIDNUM*sizeof(cfloat3),	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_RMASS,			fbuf.mf_restmass,		n*sizeof(float),				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_RDENS,			fbuf.mf_restdensity,	n*sizeof(float),				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_VISC,			fbuf.mf_visc,			n*sizeof(float),				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_VELXCOR,			fbuf.mf_velxcor,		n*sizeof(cfloat3),				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_ALPHAGRAD,		fbuf.mf_alphagrad,		n*MAX_FLUIDNUM*sizeof(cfloat3),	hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_INDICATOR,		fbuf.MFtype,			n*sizeof(int),					hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_TENSOR,			fbuf.MFtensor,		    n*sizeof(float)*9,				hipMemcpyDeviceToDevice );	
	hipMemcpy ( fbuf.msortbuf + n*BUF_TEMPTENSOR,      fbuf.MFtemptensor,		n*sizeof(float)*9,				hipMemcpyDeviceToDevice );	
	hipMemcpy ( fbuf.msortbuf + n*BUF_RTENSOR,			fbuf.MFRtensor,			n*sizeof(float)*9,				hipMemcpyDeviceToDevice );
	hipMemcpy ( fbuf.msortbuf + n*BUF_BORNID,			fbuf.MFid,				n*sizeof(int),					hipMemcpyDeviceToDevice );

	// Counting Sort - pass one, determine grid counts
	hipMemset ( fbuf.mgrid,	GRID_UCHAR,	fcuda.pnum * sizeof(int) );
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR:CopyToSortBufferCUDA: %s\n", hipGetErrorString(error) );
	} 

	CountingSortFull_ <<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );		
	hipDeviceSynchronize ();

	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR:Sorting Failed: %s\n", hipGetErrorString(error) );
	} 
	////CUDA_SAFE_CALL(hipMemcpyFromSymbol(&(fcuda.pnum), pNumFT, sizeof(int)));  //total pnum changed!
	////computeNumBlocks ( fcuda.pnum, 384, fcuda.numBlocks, fcuda.numThreads);    //threads changed!
	////fcuda.szPnts = (fcuda.numBlocks  * fcuda.numThreads);					   //szPnts changed!
	////		printf("pnum:%d,Blocknum:%d,Threadnum:%d\n",fcuda.pnum,fcuda.numBlocks,fcuda.numThreads);
	////hipDeviceSynchronize ();
}

void MpmAllocateBuffer(){

    //calculate size
    int xlen = (fcuda.maxVec - fcuda.minVec).x / fcuda.mpmSpacing;
    int ylen = (fcuda.maxVec - fcuda.minVec).y / fcuda.mpmSpacing;
    int zlen = (fcuda.maxVec - fcuda.minVec).z / fcuda.mpmSpacing;
    fcuda.mpmSize = xlen*ylen*zlen;
    fcuda.mpmXl = xlen;
    fcuda.mpmYl = ylen;
    fcuda.mpmZl = zlen;

    int splitnum = 2;

    computeNumBlocks ( fcuda.mpmSize, 384, fcuda.mpmBlocks, fcuda.mpmThreads);

    hipMalloc(&fbuf.mpmMass,   fcuda.mpmSize * sizeof(float) * splitnum);
    hipMalloc(&fbuf.mpmVel,    fcuda.mpmSize * sizeof(cfloat3) * splitnum); //0-solid, 1-fluid
    
    hipMalloc(&fbuf.mpmAlpha, fcuda.mpmSize * sizeof(float) * MAX_FLUIDNUM);
    hipMalloc(&fbuf.mpmForce, fcuda.mpmSize * sizeof(cfloat3) * splitnum);

    hipMalloc(&fbuf.mpmTensor, fcuda.mpmSize * sizeof(float) * 9);
    hipMalloc(&fbuf.mpmGid,    fcuda.mpmSize * sizeof(uint));
    hipMalloc(&fbuf.mpmGridVList, fcuda.mpmSize * sizeof(uint));
    hipMalloc(&fbuf.mpmIdSort, fcuda.mpmSize * sizeof(uint));
    hipMalloc(&fbuf.mpmGridCnt, fcuda.szGrid * sizeof(uint));
    hipMalloc(&fbuf.mpmGridOff, fcuda.szGrid * sizeof(uint));

    hipMalloc(&fbuf.mpmPos,    fcuda.mpmSize * sizeof(cfloat3));
}

void MpmSortGridCuda(){
    //initMpm <<< fcuda.mpmBlocks, fcuda.mpmThreads >>> (fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();

	thrust::device_ptr<uint> dev_keysg ( fbuf.mpmGid );
	thrust::device_ptr<uint> dev_valuesg ( fbuf.mpmIdSort );
	thrust::sort_by_key ( dev_keysg, dev_keysg + fcuda.mpmSize, dev_valuesg );
	hipDeviceSynchronize ();
	
    //MpmCalcFirstCnt << < fcuda.mpmBlocks, fcuda.mpmThreads >> > (fbuf, fcuda.mpmSize);
	hipDeviceSynchronize ();

	//MpmGetCnt << <fcuda.mpmBlocks, fcuda.mpmThreads >> > (fbuf, fcuda.mpmSize);
	hipDeviceSynchronize ();
}


void initSPH(float* restdensity,int* mftype){
	
    //initDensity<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();

	/*CUDA_SAFE_CALL( hipMemcpy( restdensity, fbuf.mf_restdensity, fcuda.pnum*sizeof(float), hipMemcpyDeviceToHost));
	CUDA_SAFE_CALL( hipMemcpy( mftype,      fbuf.MFtype,         fcuda.pnum*sizeof(int),   hipMemcpyDeviceToHost));

	double sum=0;
	int cnt=0;
	for(int i=0; i<fcuda.pnum; i++){
		if( mftype[i]==1){
			sum += restdensity[i];
			cnt++;
		}
	}
	if (cnt > 0)
		sum /= cnt;
	CUDA_SAFE_CALL( hipMemcpy( fbuf.mf_restdensity,restdensity, fcuda.pnum*sizeof(float), hipMemcpyHostToDevice));
	printf("average density %f\n",sum);*/
    
    //MpmSortGridCuda();

    //initMpm<<<fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    //hipDeviceSynchronize();	
}

void MfComputePressureCUDA ()
{
	
	/*mfFindNearest<<< fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfFindNearestVelCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();*/
	
	//ComputeBoundaryVolume <<< fcuda.numBlocks, fcuda.numThreads>>> (fbuf, fcuda.pnum);
	//hipDeviceSynchronize();

	ComputeDensityPressure <<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
	hipDeviceSynchronize();
	
	error = hipGetLastError();
	if(error != hipSuccess){
		printf("%s\n", hipGetErrorString (error));
	}
}

void MfComputeDriftVelCUDA ()
{
    //mfComputeDriftVel<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
    error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfComputeDriftVelCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void MfComputeAlphaAdvanceCUDA ()
{
	//mfComputeAlphaAdvance<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
    error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfComputeAlphaAdvanceCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}
void MfComputeCorrectionCUDA ()
{
	//mfComputeCorrection<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );	
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfComputeCorrectionCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void MfAdvanceCUDA ( float time , float dt, float ss )
{
    AdvanceParticles<<< fcuda.numBlocks, fcuda.numThreads>>> ( time, dt, ss, fbuf, fcuda.pnum );	
	
	error = hipGetLastError();
	if (error != hipSuccess) {
		fprintf ( stderr, "CUDA ERROR: MfAdvanceCUDA: %s\n", hipGetErrorString(error) );
	}    
	hipDeviceSynchronize ();
}

void ComputeForceCUDA_ProjectU(){

	//pressure force, diffusion force
	//ComputeForce_projectu<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

	//ComputeSPHtensor<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputSPHtensor: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

	//AddSPHtensorForce<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	
	hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: Adding SPH tensor Force: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();
}

void MfComputeForceCUDA(){

    //SurfaceDetection <<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
    //hipDeviceSynchronize();

	ComputeForce<<< fcuda.numBlocks, fcuda.numThreads>>> ( fbuf, fcuda.pnum );
	
	error = hipGetLastError();
	if (error != hipSuccess)
		fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
	hipDeviceSynchronize ();

    //SurfaceTension<<< fcuda.numBlocks, fcuda.numThreads >>> (fbuf, fcuda.pnum);
    //error = hipGetLastError();
	//if (error != hipSuccess)
	//	fprintf ( stderr, "CUDA ERROR: MfComputeForceCUDA: %s\n", hipGetErrorString(error) );
    //hipDeviceSynchronize ();
}



void ComputeMpmForce(){
    
    //Get Grid Mass and Velocity - 1
    //GetGridMassVel <<< fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();

    //Update Particle Strain Tensor - 2
    //CalcMpmParticleTensor <<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
    hipDeviceSynchronize();

    //Update Grid Force and Velocity - 3
    //CalcMpmGridForce<<<fcuda.mpmBlocks, fcuda.mpmThreads>>>(fbuf, fcuda.mpmSize);
    hipDeviceSynchronize();

    //Update Particle Position and Velocity - 4
    //UpdateMpmParticlePos<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
    hipDeviceSynchronize();

}

//Newly updated 
void ComputeSolidTensor(){
	//Get Density
	//ComputeDensity_CUDA<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();

	//velocity gradient, Strain, Stress
	//ComputeSolidTensor_CUDA<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void ComputeSolidForce(){
	//ComputeSolidForce_CUDA<<<fcuda.numBlocks, fcuda.numThreads>>>(fbuf, fcuda.pnum);
	hipDeviceSynchronize();
}

void InitSolid(){
	hipMemset(fbuf.MFtensor, 0, sizeof(float)*9*fcuda.pnum);
	hipMemset(fbuf.accel, 0, sizeof(cfloat3)*fcuda.pnum);
}