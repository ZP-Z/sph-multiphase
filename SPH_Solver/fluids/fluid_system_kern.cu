#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <string.h>
#include <assert.h>

#include <conio.h>

//#include <GL/glut.h>
//#include <cuda_gl_interop.h>

#include "fluid_system_host.cuh"		
#include "fluid_system_kern.cuh"
//#include "radixsort.cu"						// Build in RadixSort
#include "thrust\device_vector.h"	//thrust libs
#include "thrust\sort.h"
#include "fluidMath.cuh"

__device__ ParamCarrier paramCarrier;


void CarryParam(ParamCarrier& hostCarrier){
	hipMemcpyToSymbol(HIP_SYMBOL( paramCarrier), &hostCarrier, sizeof(hostCarrier));
}

void updateParam(FluidParams* paramCPU){
    //hipMemcpyToSymbol(HIP_SYMBOL( simData), paramCPU, sizeof(FluidParams) );
}



//Sorting

//get particle index id
__global__ void InitialSort ( bufList buf, int pnum )
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;
	register cfloat3 gridMin = paramCarrier.gridmin;
	register cfloat3 gridDelta = paramCarrier.gridIdfac;
	register cint3 gridRes = paramCarrier.gridres;
	register cint3 gridScan = paramCarrier.gridres;
	gridScan.x -= 1;
	gridScan.y -= 1;
	gridScan.z -= 1;

	register int		gs;
	register cfloat3		gcf;
	register cint3		gc;

	gcf = (buf.displayBuffer[i].pos - gridMin) * gridDelta; 
	gc = cint3( int(gcf.x), int(gcf.y), int(gcf.z) );
	
	gs = (gc.y * gridRes.z + gc.z)*gridRes.x + gc.x;
	if ( gc.x >= 1 && gc.x < gridScan.x && gc.y >= 1 && gc.y < gridScan.y && gc.z >= 1 && gc.z < gridScan.z ) {
		buf.mgcell[i] = gs;											// Grid cell insert.
		buf.midsort[i] = i;
	} else {
		buf.mgcell[i] = GRID_UNDEF;
		buf.midsort[i] = i;
	}

}

//markup the head and tail of each cell
__global__ void CalcFirstCnt ( bufList buf, int pnum )
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i>=pnum) return;
	if ((i == 0 || buf.mgcell[i]!=buf.mgcell[i-1]))
	{
		if (buf.mgcell[i]!=GRID_UNDEF)
			buf.mgridoff[buf.mgcell[i]] = i; //head id - gridoff
	}
	
	if (i!=0 && buf.mgcell[i]!=buf.mgcell[i-1] && buf.mgcell[i-1]!=GRID_UNDEF)
		buf.mgridcnt[buf.mgcell[i-1]] = i; //tail id + 1 - gridcnt
	
	if (i == pnum-1 && buf.mgcell[i]!=GRID_UNDEF)
		buf.mgridcnt[buf.mgcell[i]] = i + 1;	
}

__global__ void GetCnt ( bufList buf, int pnum )
{
	uint i = blockIdx.x * blockDim.x + threadIdx.x;		// particle index
	if (i>=pnum) return ;
	if (buf.mgcell[i]!=GRID_UNDEF)
	{
		buf.mgndx[i] = i - buf.mgridoff[buf.mgcell[i]];

		if (buf.mgndx[i] == 0) // first particle of the grid, once
			buf.mgridcnt[buf.mgcell[i]] -= buf.mgridoff[buf.mgcell[i]]; //cnt = tail - head
	}
}

//deep copy sort
__global__ void RearrangeData( bufList buf, int pnum )
{
	//for each new position, find old particle and read value
	uint i = blockIdx.x * blockDim.x + threadIdx.x;		// particle index				
	if ( i >= pnum ) return;

	int exId = buf.midsort[i]; //original id
	int cell = buf.mgcell[i];
	int sort_ndx = i;
	i = exId;

	
	if ( cell != GRID_UNDEF ) {
		
		buf.MFidTable[ buf.calcBuffer[sort_ndx].bornid ] = sort_ndx;
		
		buf.displayBuffer[sort_ndx] = *(displayPack*)(buf.msortbuf+pnum*BUF_DISPLAYBUF+i*sizeof(displayPack));
		buf.calcBuffer[sort_ndx] = *(calculationPack*)(buf.msortbuf+pnum*BUF_CALCBUF+i*sizeof(calculationPack));
	}
	else{
		buf.mgcell[sort_ndx] = GRID_UNDEF;
		buf.displayBuffer[sort_ndx].pos.Set(-1000,-1000,-1000);
	}
}

//__device__ void findNearest(int i, float& mindis, int cell, bufList buf)
//{
//	cfloat3 dist;
//	float dsq;
//	cfloat3 p = buf.displayBuffer[i].pos;
//
//	register float d2 = paramCarrier.simscale * paramCarrier.simscale;
//	register float r2 = simData.r2/d2;
//	//int j;
//
//	if (buf.mgridcnt[cell] == 0) return;
//
//	int cfirst = buf.mgridoff[cell];
//	int clast = cfirst + buf.mgridcnt[cell];
//	for (int j = cfirst; j < clast; j++) {
//		//j = buf.mgrid[cndx];
//		if (buf.displayBuffer[i].type != TYPE_BOUNDARY) {
//			dist = p - buf.displayBuffer[j].pos;
//			dsq = dot(dist, dist);
//
//			if (dsq < r2 && dsq > 0.0 && dsq*d2<mindis)
//			{
//				mindis = dsq*d2;
//				buf.midsort[i] = j;
//			}
//		}
//	}
//
//	return;
//}
//__global__ void mfFindNearest (bufList buf,int pnum)
//{
//	uint i = blockIdx.x * blockDim.x + threadIdx.x;				
//	if ( i >= pnum ) return;
//	
//	// Get search cell
//	uint gc = buf.mgcell[i];
//	if ( gc == GRID_UNDEF ) return;
//	
//	// Sum Pressures
//	cfloat3 pos = buf.displayBuffer[i].pos;
//	float mindis = 65535;
//
//	if (buf.displayBuffer[i].type == TYPE_BOUNDARY) //boundary particles
//	{
//		buf.midsort[i] = i;
//		buf.calcBuffer[i].mass = simData.pmass;
//		for (int c = 0; c<simData.gridAdjCnt; c++)
//		{
//			findNearest(i,mindis,gc+simData.gridAdj[c],buf);
//		}
//		
//		///nearest id ---> idsort
//
//		if (buf.midsort[i]!=i)
//			buf.calcBuffer[i].mass  = buf.calcBuffer[buf.midsort[i]].mass;
//
//	}
//}

__device__ void contributeDensity_boundary(uint i, float& res, uint cell, bufList buf) {
	cfloat3 dist;
	cfloat3 p = buf.displayBuffer[i].pos;
	float dsq, c;
	float massj;
	float r2 = paramCarrier.smoothradius;
	r2 = r2*r2;

	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];

	for (int j = cfirst; j < clast; j++) {

		if ( buf.displayBuffer[j].type != TYPE_BOUNDARY )
			continue;

		dist = p - buf.displayBuffer[j].pos;
		dist = dist * paramCarrier.simscale;

		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if (dsq < r2) {
			c = r2 - dsq;
			res += c * c * c ;//* buf.mf_restmass[j];
		}
	}
	return;
}

__device__ void contributeDensity (uint i, float& res, uint cell, bufList buf){
	cfloat3 dist;
	cfloat3 p = buf.displayBuffer[i].pos;
	float dsq, c;
	float massj;
	float r2 = paramCarrier.smoothradius;
	r2 = r2*r2;

	//int j;

	if ( buf.mgridcnt[cell] == 0 )
		return;

	int cfirst = buf.mgridoff[ cell ];
	int clast = cfirst + buf.mgridcnt[ cell ];

	for ( int j = cfirst; j < clast; j++ ) {
		
        if ( buf.displayBuffer[j].type==0) //fluid
			massj = buf.calcBuffer[j].mass;
		else if(buf.displayBuffer[j].type==1) //boundary
			massj = buf.calcBuffer[i].restdens * buf.calcBuffer[j].dens; // fluid density * boundary volume

		dist = p - buf.displayBuffer[j].pos;
		dist = dist * paramCarrier.simscale;

		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		if ( dsq < r2 ) {
			c = r2 - dsq;
			res += c * c * c * massj;	
		} 
	}
	return;
}

__global__ void ComputeBoundaryVolume(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if (i >= pnum) return;

	if(buf.displayBuffer[i].type != 1) //not boundary
		return;

	// Get search cell
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;						// particle out-of-range
	

	float sum = 0.0;

	//Get Boundary Density
	
	for (int c=0; c < paramCarrier.neighbornum; c++) {
		contributeDensity_boundary(i, sum, gc + paramCarrier.neighborid[c], buf);
	}

	sum = sum * paramCarrier.kpoly6;

	if (sum == 0.0) {
		printf("boundary density zero error.\n");
		sum = 1.0;
	}
		
	buf.calcBuffer[i].dens = 1 / sum ; //actually the volume
}

__global__ void ComputeDensityPressure(bufList buf,int pnum){
	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if ( i >= pnum ) return;

	if(buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;

	// Get search cell
	uint gc = buf.mgcell[i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	

	float sum = 0.0;
	float dens;
	
	dens = buf.calcBuffer[i].restdens;

	//Get Fluid Density
	for (int c=0; c < paramCarrier.neighbornum; c++) {
		contributeDensity(i, sum, gc + paramCarrier.neighborid[c], buf);
	}

	sum = sum * paramCarrier.kpoly6;
	
	if ( sum == 0.0 )
		sum = 1.0;
	
	buf.calcBuffer[i].dens = 1/sum;

	//buf.mpress[i] = ( sum - dens ) * simData.pintstiff;
	//buf.mpress[i] = (pow( sum/dens,7.0f )-1) * simData.pintstiff;
    buf.calcBuffer[i].pressure = 2.5 * dens * (pow(sum / dens, 7.0f) - 1);

    if (buf.calcBuffer[i].pressure<0)
        buf.calcBuffer[i].pressure = 0;
	//if(buf.calcBuffer[i].bornid == 0)
	//	printf("%f\n",buf.calcBuffer[i].pressure);

	//buf.mclr[i] = COLORA(1, 1-buf.mpress[i]/1000, 1-buf.mpress[i]/1000, 1);
}






__device__ cfloat3 contributeForce_new(int i, cfloat3 ipos, cfloat3 iveleval, float ipress, float idens, int cell, bufList buf, cfloat3* ivelxcor, float ivisc)
{
	//Force here represents the acceleration
	float dsq, c;
	float r2 = paramCarrier.smoothradius;
	r2 = r2*r2;

	cfloat3 dist, vmr;
	float cmterm, cmterm1;

	float pmterm, vmterm;

	int j;
	float aveDenij, cx, xterm;

	if (buf.mgridcnt[cell] == 0) return cfloat3(0, 0, 0);

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];

	cfloat3 force = cfloat3(0, 0, 0);

	for (int j = cfirst; j < clast; j++) {
		

		dist = (ipos - buf.displayBuffer[j].pos);
		dist *= paramCarrier.simscale;
		dsq = (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z);
		
		if (!(dsq < r2 && dsq > 0))
			continue;

		cx = (r2-dsq);
		dsq = sqrt(dsq);
		c = (paramCarrier.smoothradius - dsq);

		cmterm1 = paramCarrier.kspikydiff * c * c / dsq; //nabla W
		cmterm = cmterm1 * buf.calcBuffer[j].mass * buf.calcBuffer[j].dens;


		if (buf.displayBuffer[j].type == 0) 
		{
			//pressure
			pmterm = - 0.5* cmterm * (buf.calcBuffer[j].pressure) *idens;
			force += dist * pmterm;

			//viscosity
			vmr = iveleval - buf.calcBuffer[j].vel;
			vmterm = cmterm * (ivisc+buf.calcBuffer[j].visc) * idens;
			force += vmr*vmterm;

		}
		else if(buf.displayBuffer[j].type == 1){
			//pressure
			pmterm = - cmterm1 * buf.calcBuffer[i].restdens * buf.calcBuffer[j].dens *  (ipress ) *idens*idens;
			force += dist * pmterm;
			
			//artificial boundary viscosity			
			vmr = iveleval - buf.calcBuffer[j].vel;
			float pi_ij = vmr.x*dist.x + vmr.y*dist.y + vmr.z*dist.z;
			if (pi_ij < 0) {
				pi_ij = pi_ij / (dist.x*dist.x + dist.y*dist.y + dist.z*dist.z + r2 * 0.01);
				pi_ij = pi_ij * paramCarrier.bvisc * paramCarrier.smoothradius  /2 / buf.calcBuffer[i].restdens ;
				pi_ij = cmterm1 * buf.calcBuffer[i].restdens * buf.calcBuffer[j].dens * pi_ij;
				force += (dist * pi_ij);
			}
		}
		
	}
	return force;
}

__global__ void ComputeForce (bufList buf, int pnum){
	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if ( i >= pnum)
		return;
	if(buf.displayBuffer[i].type==1)
		return;

	// Get search cell
	uint gc = buf.mgcell[i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	

	register cfloat3 ipos = buf.displayBuffer[i].pos;
	register cfloat3 iveleval = buf.calcBuffer[i].vel;
	register float ipress = buf.calcBuffer[i].pressure;
	register float idens = buf.calcBuffer[i].dens;
	//register cfloat3 ivelxcor = buf.calcBuffer[i].velxcor;
	register float ivisc = buf.calcBuffer[i].visc;

	register cfloat3 force = cfloat3(0,0,0);	
	cfloat3 ivelxcor = cfloat3(0,0,0);

	for (int c=0; c < paramCarrier.neighbornum; c++) {
		force += contributeForce_new ( i,  ipos, iveleval, ipress, idens, gc + paramCarrier.neighborid[c], buf, &ivelxcor, ivisc);
	}
	buf.calcBuffer[i].accel = force;

}

__global__ void AdvanceParticles(bufList buf, int numPnts)
{

	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if (i >= numPnts) return;
	if (buf.displayBuffer[i].type==1)
		return;

	if (buf.mgcell[i] == GRID_UNDEF) {
		buf.displayBuffer[i].pos = cfloat3(-1000, -1000, -1000);
		buf.calcBuffer[i].vel = cfloat3(0, 0, 0);
		return;
	}

	// Get particle vars
	cfloat3 accel, norm;
	float diff, adj, accelLen;
	cfloat3 pos = buf.displayBuffer[i].pos;
	cfloat3 veval = buf.calcBuffer[i].vel;
	float dt = paramCarrier.dt;
	float ss = paramCarrier.simscale;

	accel = cfloat3(0, 0, 0);
	
	/*
	// Soft Boundaries
	// Y-axis
	diff = simData.pradius - (pos.y - (simData.pboundmin.y + (pos.x-simData.pboundmin.x)*simData.pground_slope)) * ss;
	float slope = 0;
	if (diff > EPSILON) {
		norm = cfloat3(-slope, 1.0 -slope, 0);
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;
		//norm = -cfloat3(vel.x, 0, vel.z);
		//accel += norm*10;

	}

	diff = simData.pradius - (simData.pboundmax.y - pos.y)*ss;
	if (diff > EPSILON) {
		norm = cfloat3(0, -1, 0);
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;
	}

	// X-axis
	diff = simData.pradius - (pos.x - (simData.pboundmin.x + (sin(time*simData.pforce_freq)+1)*0.5 * simData.pforce_min))*ss;
	if (diff > EPSILON) {
		norm = cfloat3(1, 0, 0);
		adj = (simData.pforce_min+1) * simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;//*scale_dens;
	}

	diff = simData.pradius - ((simData.pboundmax.x - (sin(time*simData.pforce_freq)+1)*0.5*simData.pforce_max) - pos.x)*ss;
	if (diff > EPSILON) {
		norm = cfloat3(-1, 0, 0);
		adj = (simData.pforce_max+1) * simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;//*scale_dens;
	}

	// Z-axis
	diff = simData.pradius - (pos.z - simData.pboundmin.z) * ss;
	if (diff > EPSILON) {
		norm = cfloat3(0, 0, 1);
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;//*scale_dens;
	}

	diff = simData.pradius - (simData.pboundmax.z - pos.z)*ss;
	if (diff > EPSILON) {
		norm = cfloat3(0, 0, -1);
		adj = simData.pextstiff * diff - simData.pdamp * dot(norm, veval);
		norm *= adj; accel += norm;//*scale_dens;
	}
	*/
	//End Soft Boundary

	accel += buf.calcBuffer[i].accel;
	accel += paramCarrier.gravity;

	// Accel Limit
	//accelLen = accel.x*accel.x + accel.y*accel.y + accel.z*accel.z;
	//if (accelLen > simData.AL2) {
	//	accel *= simData.AL / sqrt(accelLen);
	//}

	cfloat3 vel = buf.calcBuffer[i].vel;
	//Velocity Limit
	/*float vmod = sqrtf(dot(vel, vel));
	if (vmod > simData.fluidVConstraint)
	vel *= simData.fluidVConstraint / vmod;*/

	cfloat3 vnext = accel*dt + vel;		// v(t+1/2) = v(t-1/2) + a(t) dt
	buf.displayBuffer[i].pos += vnext*dt/ss;
	buf.calcBuffer[i].veleval = (vel+vnext)*0.5;
	buf.calcBuffer[i].vel = vnext;

	//For Drift Velocity Calculation of Next Step
	//buf.mforce[i] = simData.pgravity - accel;
	//buf.mforce[i] = accel;
	//buf.mforce[i] = cfloat3(0, 0, 0);

	//Color Setting
	/*if(buf.MFtype[i]==3)
	buf.mclr[i] = COLORA(buf.mf_alpha[i*MAX_FLUIDNUM+2],buf.mf_alpha[i*MAX_FLUIDNUM+1],buf.mf_alpha[i*MAX_FLUIDNUM+0],0.6);
	else
	buf.mclr[i] = COLORA(buf.mf_alpha[i*MAX_FLUIDNUM+2],buf.mf_alpha[i*MAX_FLUIDNUM+1],buf.mf_alpha[i*MAX_FLUIDNUM+0],1);*/

}





/*-------------------------

	Surface Tension - Yang Tao 

--------------------------*/



__device__ void contributeCover(int i, int cell, bufList buf){

}

__global__ void SurfaceDetection(bufList buf, int pnum){
    uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if ( i >= pnum)
		return;
	
	// Get search cell
	uint gc = buf.mgcell[i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	

	for (int c=0; c < paramCarrier.neighbornum; c++) {
		contributeCover ( i, gc + paramCarrier.neighborid[c], buf );
	}
}



__device__ void contributeSurfaceTensionYT(int i, cfloat3& res, int cell, bufList buf)
{
    //Force here represents the acceleration
	float dsq, c;
	float sr2 = paramCarrier.smoothradius;
	sr2 *= sr2;
	cfloat3 dist;		
	int j;	

	if ( buf.mgridcnt[cell] == 0 )
        return;	

	int cfirst = buf.mgridoff[ cell ];
	int clast = cfirst + buf.mgridcnt[ cell ];

    float tensionRadius2 = paramCarrier.surfaceTensionK * paramCarrier.surfaceTensionK * sr2;
    float tR = sqrt(tensionRadius2);

    float forceMod;
    float ffTensionC = paramCarrier.surfaceTensionFluidC;

	for ( int j = cfirst; j < clast; j++ ) {	

        dist = buf.displayBuffer[i].pos - buf.displayBuffer[j].pos;
        dist = dist * paramCarrier.simscale;
		dsq = dot(dist,dist);
        if (dsq != 0.0 && dsq <= tR){
            dsq = sqrt(dsq);
            forceMod = ffTensionC * cos( 1.5 * 3.14159 * dsq/ tR) / dsq;
            
            res += dist*forceMod;
        }

    }
}

__global__ void SurfaceTension(bufList buf, int pnum){
    uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if ( i >= pnum)
		return;
	//if(buf.displayBuffer[i].type==1)
	//	return;

	// Get search cell
	uint gc = buf.mgcell[i];
	if ( gc == GRID_UNDEF ) return;						// particle out-of-range
	

	register cfloat3 force = cfloat3(0,0,0);	

	for (int c=0; c < paramCarrier.neighbornum; c++) {
		contributeSurfaceTensionYT ( i, force, gc + paramCarrier.neighborid[c], buf);
	}

	buf.calcBuffer[i].accel += force;

}


/*-------------------------

	End Surface Tension

--------------------------*/

__device__ void contributeAcc_adv(int i, cfloat3& acc_adv, int cell, bufList buf) {
	
	if (buf.mgridcnt[cell] == 0)
		return;
	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	float c;
	float h = paramCarrier.smoothradius;
	float h2 = h*h;

	cfloat3 xij, vij;
	float dist;
	int j;

	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 ivel = buf.calcBuffer[i].vel;
	float ivisc = buf.calcBuffer[i].visc;
	float idens = buf.calcBuffer[i].dens;
	float cmterm;
	float nW_fac;
	float vmterm;
	cfloat3 accel = cfloat3(0, 0, 0);

	for (int j = cfirst; j < clast; j++) {

		xij = ipos - buf.displayBuffer[j].pos;
		xij *= paramCarrier.simscale;
		dist = sqrt(dot(xij,xij));

		if (!(dist < h && dist > 0))
			continue;
		c = h - dist;

		nW_fac = paramCarrier.kspikydiff * c * c / dist;
		cmterm = nW_fac * buf.calcBuffer[j].mass * buf.calcBuffer[j].dens;

		if (buf.displayBuffer[j].type == 0)
		{
			vij = ivel - buf.calcBuffer[j].vel;
			vmterm = cmterm * (ivisc+buf.calcBuffer[j].visc) * idens;
			accel += vij*vmterm;
		}
		else if (buf.displayBuffer[j].type == 1) {

			//artificial boundary viscosity			
			vij = ivel - buf.calcBuffer[j].vel;
			float pi_ij = dot(xij,vij);
			if (pi_ij < 0) {
				pi_ij = pi_ij / (dist*dist + h2 * 0.01);
				pi_ij = pi_ij * paramCarrier.bvisc * paramCarrier.smoothradius  /2 / buf.calcBuffer[i].restdens;
				pi_ij = nW_fac * buf.calcBuffer[i].restdens * buf.calcBuffer[j].dens * pi_ij;
				accel += (xij * pi_ij);
			}
		}

	}
	
	acc_adv = acc_adv + accel;
}

__device__ void contributeDii(int i, cfloat3& dii, int cell, bufList buf) {
	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 jpos,xij;
	float dist;
	float h = paramCarrier.smoothradius;
	float c;
	float nW_fac;
	cfloat3 nablaWij;

	for (int j = cfirst; j < clast; j++) {
		
		jpos = buf.displayBuffer[j].pos;
		xij = ipos - jpos;
		xij = xij*paramCarrier.simscale;
		dist = sqrt(dot(xij,xij));
		if(!(dist<h && dist>0))
			continue;
		
		c = h - dist;
		nW_fac = paramCarrier.kspikydiff * c * c / dist;
		nablaWij = xij * nW_fac;
		
		if (buf.displayBuffer[j].type!=TYPE_BOUNDARY)
			dii = dii - nablaWij*buf.calcBuffer[j].mass;
		else
			dii = dii - nablaWij*buf.calcBuffer[i].restdens*buf.calcBuffer[j].dens;
	}
}

__global__ void ComputeDii(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;			
	if (i >= pnum) return;
	if (buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;



	//v_adv
	cfloat3 acc_adv(0,0,0);
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeAcc_adv(i, acc_adv, gc+paramCarrier.neighborid[c],buf);
	}
	acc_adv += paramCarrier.gravity;
	buf.calcBuffer[i].veleval = buf.calcBuffer[i].vel + acc_adv * paramCarrier.dt;



	//dii
	cfloat3 dii(0,0,0);
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeDii(i,dii,gc+paramCarrier.neighborid[c],buf);
	}
	
	dii = dii * buf.calcBuffer[i].dens * buf.calcBuffer[i].dens * paramCarrier.dt * paramCarrier.dt;
	//if (i%100==0)
	//	printf("dii: %.20f %.20f %.20f\n", dii.x, dii.y, dii.z);

	buf.dii[i] = dii;
	
}


__device__ void contributeDensChange(int i, float& denschange, int cell, bufList buf) {
	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 ivel = buf.calcBuffer[i].veleval;
	cfloat3 jpos, xij, vij;
	float dist;
	float h = paramCarrier.smoothradius;
	float c;
	float nW_fac;
	cfloat3 nablaWij;
	float sum = 0;

	for (int j = cfirst; j < clast; j++) {
		jpos = buf.displayBuffer[j].pos;
		xij = ipos - jpos;
		xij = xij*paramCarrier.simscale;
		dist = sqrt(dot(xij,xij));
		if (!(dist<h && dist>0))
			continue;

		vij = ivel - buf.calcBuffer[j].veleval;
		c = h - dist;
		nW_fac = paramCarrier.kspikydiff * c * c / dist; //nabla W
		nablaWij = xij*nW_fac;

		if(buf.displayBuffer[j].type!=TYPE_BOUNDARY)
			sum += dot(vij, nablaWij) * buf.calcBuffer[j].mass * paramCarrier.dt;
		else
			sum += dot(vij, nablaWij) * buf.calcBuffer[j].dens * buf.calcBuffer[i].restdens * paramCarrier.dt;
	}
	denschange += sum;
}

__device__ void contributeAii(int i, float& aii, int cell, bufList buf) {
	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 ivel = buf.calcBuffer[i].veleval;
	cfloat3 jpos, xij;
	float dist;
	float h = paramCarrier.smoothradius;
	float c;
	float nW_fac;
	cfloat3 nablaWij;

	float sum = 0;
	cfloat3 dji;
	cfloat3 dii = buf.dii[i];
	cfloat3 dtmp;

	for (int j = cfirst; j < clast; j++) {
		if (buf.displayBuffer[j].type==TYPE_BOUNDARY)
			continue;
		jpos = buf.displayBuffer[j].pos;
		xij = ipos - jpos;
		xij = xij*paramCarrier.simscale;
		dist = sqrt(dot(xij,xij));

		if (!(dist<h && dist>0))
			continue;
		c = h - dist;
		nW_fac = paramCarrier.kspikydiff * c * c / dist; //nabla W
		nablaWij = xij*nW_fac;

		dji = nablaWij * buf.calcBuffer[i].mass* buf.calcBuffer[i].dens * buf.calcBuffer[i].dens * paramCarrier.dt
			* paramCarrier.dt;
		dtmp = dii - dji;
		sum += dot(dtmp, nablaWij) * buf.calcBuffer[j].mass;
	}
	aii += sum;

}

__global__ void ComputeAii(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= pnum) return;
	if (buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;

	//rho_i_adv
	float denschange = 0;
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeDensChange(i, denschange, gc+paramCarrier.neighborid[c],buf);
	}
	buf.rho_adv[i] = 1/buf.calcBuffer[i].dens + denschange;

	//p_i_l0
	buf.press_l[i] = 0.5 * buf.calcBuffer[i].pressure;

	//aii
	float aii = 0;
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeAii(i, aii, gc+paramCarrier.neighborid[c], buf);
	}
	buf.aii[i] = aii;

}

__device__ void contributeDP(int i, cfloat3& dp, int cell, bufList buf) {
	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 jpos, xij;
	float dist;
	float h = paramCarrier.smoothradius;
	float c;
	float nW_fac;
	cfloat3 nablaWij;

	for (int j = cfirst; j < clast; j++) {
		if (buf.displayBuffer[j].type==TYPE_BOUNDARY)
			continue;
		jpos = buf.displayBuffer[j].pos;
		xij = ipos - jpos;
		xij = xij*paramCarrier.simscale;
		dist = sqrt(xij.x*xij.x + xij.y*xij.y + xij.z*xij.z);
		if (!(dist<h && dist>0))
			continue;
		c = h - dist;
		nW_fac = paramCarrier.kspikydiff * c * c / dist; //nabla W
		nablaWij = xij*nW_fac;
		
		dp = dp + nablaWij * buf.press_l[j] * buf.calcBuffer[j].mass * buf.calcBuffer[j].dens
			* buf.calcBuffer[j].dens * (-1);
	}

}

__global__ void Pressure_DP(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= pnum) return;
	if (buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;

	cfloat3 dp(0,0,0);
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeDP(i,dp,gc+paramCarrier.neighborid[c],buf);
	}
	dp = dp * paramCarrier.dt * paramCarrier.dt;
	buf.dijpj[i] = dp;
}

__device__ void contributeP_updateterm(int i, float& updateterm, int cell, bufList buf) {
	if (buf.mgridcnt[cell] == 0)
		return;

	int cfirst = buf.mgridoff[cell];
	int clast = cfirst + buf.mgridcnt[cell];
	cfloat3 ipos = buf.displayBuffer[i].pos;
	cfloat3 jpos, xij;
	float dist;
	float h = paramCarrier.smoothradius;
	float c;
	float nW_fac;
	cfloat3 nablaWij;
	
	cfloat3 dijpj = buf.dijpj[i];
	cfloat3 djj;
	cfloat3 djkpk, dji;
	cfloat3 tmpf3;

	for (int j = cfirst; j < clast; j++) {
		
		jpos = buf.displayBuffer[j].pos;
		xij = ipos - jpos;
		xij = xij*paramCarrier.simscale;
		dist = sqrt(xij.x*xij.x + xij.y*xij.y + xij.z*xij.z);
		if (!(dist<h && dist>0))
			continue;
		c = h - dist;
		nW_fac = paramCarrier.kspikydiff * c * c / dist; //nabla W
		nablaWij = xij*nW_fac;

		if (buf.displayBuffer[j].type!=TYPE_BOUNDARY) {
			djj = buf.dii[j];
			dji = nablaWij*buf.calcBuffer[i].mass*buf.calcBuffer[i].dens*buf.calcBuffer[i].dens*
				paramCarrier.dt * paramCarrier.dt;
			djkpk = buf.dijpj[j] - dji * buf.press_l[i];
			tmpf3 = dijpj - djj*buf.press_l[j] - djkpk;
			updateterm += buf.calcBuffer[j].mass * dot(tmpf3, nablaWij);
		}
		else {
			updateterm += buf.calcBuffer[i].restdens * buf.calcBuffer[j].dens * dot(dijpj,nablaWij);
		}
		
	}

}


__global__ void Pressure_Iter(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= pnum) return;
	if (buf.displayBuffer[i].type==TYPE_BOUNDARY){
		buf.densityResidue[i] = 0;
		return;
	}
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;

	//compute update term
	float updateterm = 0;
	for (int c=0; c<paramCarrier.neighbornum; c++) {
		contributeP_updateterm(i, updateterm, gc+paramCarrier.neighborid[c], buf);
	}

	//update pressure
	float omega = 0.3;
	if (abs(buf.aii[i])<EPSILON) {
		buf.press_l1[i] = buf.press_l[i];
	}
	else{
		buf.press_l1[i] = (1-omega)*buf.press_l[i] + omega/buf.aii[i]
			*(buf.calcBuffer[i].restdens - buf.rho_adv[i] - updateterm); //<-- residue
	}
	
	if(buf.press_l1[i]<0)
		buf.press_l1[i]=0;

	float rhoil = buf.rho_adv[i]+updateterm+buf.aii[i]*buf.press_l[i];
	
	buf.densityResidue[i] = -buf.calcBuffer[i].restdens+(buf.rho_adv[i]+updateterm+buf.aii[i]*buf.press_l[i]);
	
	cfloat3 force(0,0,0);
	force = buf.dii[i]*buf.press_l[i] + buf.dijpj[i];
	//if (i%100==0) {
	//	printf("%f %f %f %f\n",force.x,force.y,force.z,buf.press_l[i]);
	//}
	buf.calcBuffer[i].accel = force/paramCarrier.dt /paramCarrier.dt;
	buf.calcBuffer[i].pressure = buf.press_l1[i];
}

__global__ void IntegrateIISPH(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= pnum) return;
	if (buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;

	buf.calcBuffer[i].vel = buf.calcBuffer[i].veleval + buf.calcBuffer[i].accel * paramCarrier.dt;
	
	buf.displayBuffer[i].pos += buf.calcBuffer[i].vel * paramCarrier.dt/paramCarrier.simscale;
	//buf.calcBuffer[i].vel.Set(0,0,0);
}

__global__ void ComputeDensityIISPH(bufList buf, int pnum) {
	uint i = blockIdx.x * blockDim.x + threadIdx.x;	// particle index				
	if (i >= pnum) return;

	if (buf.displayBuffer[i].type==TYPE_BOUNDARY)
		return;

	// Get search cell
	uint gc = buf.mgcell[i];
	if (gc == GRID_UNDEF) return;						// particle out-of-range


	float sum = 0.0;
	float dens;

	dens = buf.calcBuffer[i].restdens;

	//Get Fluid Density
	for (int c=0; c < paramCarrier.neighbornum; c++) {
		contributeDensity(i, sum, gc + paramCarrier.neighborid[c], buf);
	}

	sum = sum * paramCarrier.kpoly6;

	if (sum == 0.0)
		sum = 1.0;

	buf.calcBuffer[i].dens = 1/sum;
}